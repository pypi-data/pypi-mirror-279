#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <random>

#include "common/check.h"
#include "variable/impl/det_variable.h"

namespace sok {

template <typename T>
std::vector<size_t> argsort(const std::vector<T>& array) {
  std::vector<size_t> indices(array.size());
  std::iota(indices.begin(), indices.end(), 0);
  std::sort(indices.begin(), indices.end(), [&array](size_t left, size_t right) -> bool {
    // sort indices according to corresponding array element
    return array[left] < array[right];
  });

  return indices;
}

template <typename KeyType, typename ValueType>
void gather_by_index(std::vector<size_t>& inx, KeyType* src_key, ValueType* src_value,
                     KeyType* dst_key, ValueType* dst_value, size_t num_keys, size_t value_dim) {
  for (int i = 0; i < num_keys; i++) {
    dst_key[i] = src_key[inx[i]];
    std::memcpy(dst_value + i * value_dim, src_value + (inx[i]) * value_dim,
                sizeof(ValueType) * value_dim);
  }
  return;
}

// TODO: Move this into cuco::initializer
__global__ static void setup_kernel(unsigned long long seed, hiprandState* states) {
  auto grid = cooperative_groups::this_grid();
  hiprand_init(seed, grid.thread_rank(), 0, &states[grid.thread_rank()]);
}

static void set_curand_states(hiprandState** states, hipStream_t stream = 0) {
  int device;
  CUDACHECK(hipGetDevice(&device));
  hipDeviceProp_t deviceProp;
  CUDACHECK(hipGetDeviceProperties(&deviceProp, device));
  // TODO: Use a more compatible way instead of `2048` to calculate the size.
  //       Note that the code of cuco::initializer also needs to be modified.
  // CUDACHECK(
  //     hipMallocAsync(states, sizeof(hiprandState) * deviceProp.multiProcessorCount * 2048,
  //     stream));
  CUDACHECK(hipMalloc(states, sizeof(hiprandState) * deviceProp.multiProcessorCount * 2048));
  std::random_device rd;
  auto seed = rd();
  setup_kernel<<<deviceProp.multiProcessorCount * 2, 1024, 0, stream>>>(seed, *states);
  // To avoid unexpected errors caused by using `states` in other non-blocking streams.
  // It's OK to do synchronization here because this method should be called very few times.
  // CUDACHECK(hipStreamSynchronize(stream));
}

class ConstInitializer {
 private:
  float val_;

 public:
  ConstInitializer(float val) : val_(val) {}
  // Note that the `val_` below is not the `val_` in the host object, because the entire
  // `ConstInitializer` object will be passed by value to a __global__ function before
  // this __device__ function is called.
  __device__ float operator()() const { return val_; }
};

static void parse_initializer(const std::string& initializer, bool& is_const, float& val) {
  if (initializer == "random" || initializer == "") {
    is_const = false;
    return;
  }

  is_const = true;
  if (initializer == "ones") {
    val = 1.0;
  } else if (initializer == "zeros") {
    val = 0.0;
  } else {
    try {
      val = std::stof(initializer);
    } catch (std::invalid_argument& err) {
      throw std::runtime_error("Unrecognized initializer {" + initializer + "}");
    }
  }
}

template <typename KeyType, typename ValueType>
DETVariable<KeyType, ValueType>::DETVariable(size_t dimension, size_t initial_capacity,
                                             const std::string& initializer, hipStream_t stream)
    : dimension_(dimension),
      initial_capacity_(initial_capacity),
      initializer_(initializer),
      curand_states_(nullptr) {
  if (dimension_ <= 0) {
    throw std::invalid_argument("dimension must > 0 but got " + std::to_string(dimension));
  }

  set_curand_states(&curand_states_, stream);

  bool use_const_initializer = false;
  float initial_val = 0.0;
  parse_initializer(initializer_, use_const_initializer, initial_val);

  map_ = std::make_unique<cuco::dynamic_map<KeyType, ValueType, cuco::initializer>>(
      dimension_, initial_capacity_,
      cuco::initializer(curand_states_, use_const_initializer, initial_val));
  if (!map_) {
    throw std::runtime_error("Create DET failed");
  }
  map_->initialize(stream);
}

template <typename KeyType, typename ValueType>
DETVariable<KeyType, ValueType>::~DETVariable() {
  map_->uninitialize();
  if (curand_states_) {
    CUDACHECK(hipFree(curand_states_));
  }
}

template <typename KeyType, typename ValueType>
int64_t DETVariable<KeyType, ValueType>::rows() {
  return map_->get_size();
}

template <typename KeyType, typename ValueType>
int64_t DETVariable<KeyType, ValueType>::cols() {
  return dimension_;
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::eXport(KeyType* keys, ValueType* values,
                                             hipStream_t stream) {
  size_t num_keys = rows();
  size_t dim = cols();

  // `keys` and `values` are pointers of host memory
  KeyType* d_keys;
  CUDACHECK(hipMallocManaged(&d_keys, sizeof(KeyType) * num_keys));
  ValueType* d_values;
  CUDACHECK(hipMallocManaged(&d_values, sizeof(ValueType) * num_keys * dim));

  map_->eXport(d_keys, d_values, num_keys, stream);
  CUDACHECK(hipStreamSynchronize(stream));

  std::vector<KeyType> dk_vector = std::vector<KeyType>(d_keys, d_keys + num_keys);
  auto dk_indices = argsort(dk_vector);

  gather_by_index(dk_indices, d_keys, d_values, keys, values, num_keys, dim);

  // clang-format on
  CUDACHECK(hipFree(d_keys));
  CUDACHECK(hipFree(d_values));
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::eXport_if(KeyType* keys, ValueType* values, size_t* counter,
                                                uint64_t threshold, hipStream_t stream) {
  throw std::runtime_error("SOK dynamic variable with DET backend don't support eXport_if");
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::assign(const KeyType* keys, const ValueType* values,
                                             size_t num_keys, hipStream_t stream) {
  size_t dim = cols();

  // `keys` and `values` are pointers of host memory
  KeyType* d_keys;
  CUDACHECK(hipMallocManaged(&d_keys, sizeof(KeyType) * num_keys));
  ValueType* d_values;
  CUDACHECK(hipMallocManaged(&d_values, sizeof(ValueType) * num_keys * dim));

  // clang-format off
  //CUDACHECK(hipMemcpyAsync(d_keys, keys, sizeof(KeyType) * num_keys,
  //                          hipMemcpyHostToDevice, stream));
  std::memcpy(d_keys,keys, sizeof(KeyType) * num_keys);
  // clang-format on
  map_->lookup(d_keys, d_values, num_keys, stream);
  CUDACHECK(hipStreamSynchronize(stream));

  // CUDACHECK(hipMemcpyAsync(d_values, values, sizeof(ValueType) * num_keys * dim,
  //                           hipMemcpyHostToDevice, stream));

  std::memcpy(d_values, values, sizeof(ValueType) * num_keys * dim);
  map_->scatter_update(d_keys, d_values, num_keys, stream);
  CUDACHECK(hipStreamSynchronize(stream));
  CUDACHECK(hipFree(d_keys));
  CUDACHECK(hipFree(d_values));
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::lookup(const KeyType* keys, ValueType* values,
                                             size_t num_keys, hipStream_t stream) {
  map_->lookup(keys, values, num_keys, stream);
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::lookup(const KeyType* keys, ValueType** values,
                                             size_t num_keys, hipStream_t stream) {
  map_->lookup(keys, values, num_keys, stream);
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::lookup_with_evict(const KeyType* keys, KeyType* tmp_keys,
                                                        ValueType* tmp_values, ValueType* values,
                                                        uint64_t* evict_num_keys, uint64_t num_keys,
                                                        hipStream_t stream) {
  throw std::runtime_error(
      "SOK dynamic variable with DET backend don't support lookup_with_evict!");
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::copy_evict_keys(const KeyType* keys, const ValueType* values,
                                                      size_t num_keys, size_t dim,
                                                      KeyType* ret_keys, ValueType* ret_values,
                                                      hipStream_t stream) {
  throw std::runtime_error(
      "SOK dynamic variable with DET backend don't support lookup_with_evict!");
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::scatter_add(const KeyType* keys, const ValueType* values,
                                                  size_t num_keys, hipStream_t stream) {
  map_->scatter_add(keys, values, num_keys, stream);
}

template <typename KeyType, typename ValueType>
void DETVariable<KeyType, ValueType>::scatter_update(const KeyType* keys, const ValueType* values,
                                                     size_t num_keys, hipStream_t stream) {
  map_->scatter_update(keys, values, num_keys, stream);
}

template class DETVariable<int32_t, float>;
template class DETVariable<int64_t, float>;

}  // namespace sok
