#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cassert>
#include <data_readers/multi_hot/split_batch.hpp>

namespace HugeCTR {
template <bool ISFLOAT = true>
struct DenseOp_t {
  __host__ __device__ __forceinline__ float operator()(const int* in) { return 0.f; }
  DenseOp_t() = default;
};
template <>
struct DenseOp_t<true> {
  __host__ __device__ __forceinline__ float operator()(const int* in) {
    return *reinterpret_cast<const float*>(in);
  }
};
template <>
struct DenseOp_t<false> {
  __host__ __device__ __forceinline__ float operator()(const int* in) {
    return static_cast<float>(logf(*in + 1.f));
  }
};

using int_dense_op_t = DenseOp_t<false>;
using float_dense_op_t = DenseOp_t<true>;

template <typename DenseType, typename SparseType, typename DenseOp>
__global__ void split_feat_major_kernel(float* __restrict label, int label_dim,
                                        DenseType* __restrict dense, int dense_dim,
                                        SparseType** __restrict sparse_tensors, int sparse_dim,
                                        const int* __restrict label_dense_sparse,
                                        const int* __restrict bucket_ids,
                                        const int* __restrict bucket_positions,
                                        const int* __restrict max_hotnesses, uint32_t batch_size,
                                        uint32_t sample_dim, DenseOp dop) {
  for (uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < batch_size * sample_dim;
       idx += blockDim.x * gridDim.x) {
    const uint32_t row = idx / sample_dim;
    const uint32_t col = idx - row * sample_dim;

    if (col < label_dim)  // store in label tensor
    {
      auto col_data = label_dense_sparse[idx];  // Load column
      label[row * label_dim + col] = static_cast<float>(col_data);
    } else if (col < label_dim + dense_dim)  // store in dense tensor
    {
      const auto dense_col = col - label_dim;
      // sizeof(int) == sizeof(float)
      const int* col_data = reinterpret_cast<const int*>(label_dense_sparse) + idx;
      dense[row * dense_dim + dense_col] = static_cast<DenseType>(dop(col_data));
    } else  // store in sparse tensors
    {
      auto col_data = label_dense_sparse[idx];  // Load column
      if constexpr (std::is_same<SparseType, long long>::value) {
        const auto sparse_col = col - label_dim - dense_dim;
        const auto bucket_id = bucket_ids[sparse_col / 2];
        const auto bucket_idx =
            (row * max_hotnesses[bucket_id] + bucket_positions[sparse_col / 2]) * 2 +
            (sparse_col & 1);
        reinterpret_cast<int**>(sparse_tensors)[bucket_id][bucket_idx] = col_data;
      } else {
        const auto sparse_col = col - label_dim - dense_dim;
        const auto bucket_id = bucket_ids[sparse_col];
        const auto bucket_idx = row * max_hotnesses[bucket_id] + bucket_positions[sparse_col];
        sparse_tensors[bucket_id][bucket_idx] = static_cast<SparseType>(col_data);
      }
    }
  }
}

template <typename DenseType, typename SparseType>
void split_3_way_feat_major(core23::Tensor label_tensor, core23::Tensor dense_tensor,
                            core23::Tensor sparse_tensors, core23::Tensor label_dense_sparse_tensor,
                            core23::Tensor bucket_ids, core23::Tensor bucket_positions,
                            core23::Tensor max_hotnesses, hipStream_t stream,
                            bool dense_is_float) {
  const auto batch_size = label_dense_sparse_tensor.size(0);
  const auto label_dim = label_tensor.size(1);
  const auto dense_dim = dense_tensor.size(1);
  const auto sparse_dim = sparse_tensors.size(0);
  const auto sample_dim = label_dense_sparse_tensor.size(1);
  assert(label_dim > 0 && "label_dim is 0");
  assert(dense_dim > 0 && "dense_dim is 0");
  assert(sample_dim > 0 && "sample_dim is 0");

  constexpr dim3 block_dim(128);
  const dim3 grid_dim((batch_size * sample_dim + block_dim.x - 1) / block_dim.x);
  if (dense_is_float) {
    auto DOP = float_dense_op_t();
    split_feat_major_kernel<<<grid_dim, block_dim, 0, stream>>>(
        label_tensor.data<float>(), label_dim, dense_tensor.data<DenseType>(), dense_dim,
        reinterpret_cast<SparseType**>(sparse_tensors.data()), sparse_dim,
        label_dense_sparse_tensor.data<int>(), bucket_ids.data<int>(), bucket_positions.data<int>(),
        max_hotnesses.data<int>(), batch_size, sample_dim, DOP);
  } else {
    auto DOP = int_dense_op_t();
    split_feat_major_kernel<<<grid_dim, block_dim, 0, stream>>>(
        label_tensor.data<float>(), label_dim, dense_tensor.data<DenseType>(), dense_dim,
        reinterpret_cast<SparseType**>(sparse_tensors.data()), sparse_dim,
        label_dense_sparse_tensor.data<int>(), bucket_ids.data<int>(), bucket_positions.data<int>(),
        max_hotnesses.data<int>(), batch_size, sample_dim, DOP);
  }

  HCTR_LIB_THROW(hipPeekAtLastError());
}

#define INSTANTIATE_SPLIT_3_WAY_23(DENSE_T, SPARSE_T)                                          \
  template void split_3_way_feat_major<DENSE_T, SPARSE_T>(                                     \
      core23::Tensor label_tensor, core23::Tensor dense_tensor, core23::Tensor sparse_tensors, \
      core23::Tensor label_dense_sparse_tensor, core23::Tensor bucket_ids,                     \
      core23::Tensor bucket_positions, core23::Tensor max_hotnesses, hipStream_t stream,      \
      bool float_dense)

INSTANTIATE_SPLIT_3_WAY_23(float, uint32_t);
INSTANTIATE_SPLIT_3_WAY_23(__half, uint32_t);
INSTANTIATE_SPLIT_3_WAY_23(float, long long);
INSTANTIATE_SPLIT_3_WAY_23(__half, long long);

}  // namespace HugeCTR
