#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common.hpp>
#include <layers/concat_layer.hpp>
#include <network_buffer_channels.hpp>
#include <utils.hpp>

namespace HugeCTR {

namespace {

template <typename T>
__global__ void concat_fwd_kernel(T* out, const int2 out_dim, T* in, const int2 in_dim,
                                  int offset) {
  for (int mi = blockIdx.x; mi < in_dim.x; mi += gridDim.x) {
    for (int ni = threadIdx.x; ni < in_dim.y; ni += blockDim.x) {
      out[mi * out_dim.y + offset + ni] = in[mi * in_dim.y + ni];
    }
  }
}

template <typename T>
__global__ void concat_bwd_kernel(T* out, const int2 out_dim, T* in, const int2 in_dim,
                                  int offset) {
  for (int mi = blockIdx.x; mi < in_dim.x; mi += gridDim.x) {
    for (int ni = threadIdx.x; ni < in_dim.y; ni += blockDim.x) {
      in[mi * in_dim.y + ni] = out[mi * out_dim.y + offset + ni];
    }
  }
}

}  // namespace

template <typename T>
ConcatLayer<T>::ConcatLayer(const std::vector<core23::Tensor>& input_tensors,
                            core23::Tensor& output_tensor,
                            const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(input_tensors, {}, gpu_resource) {
  try {
    if (input_tensors_.empty()) {
      HCTR_OWN_THROW(Error_t::WrongInput, "Empty input tensors");
    }
    size_t n_input_tensors = input_tensors_.size();
    int64_t height = 0;
    int64_t new_width = 0;
    for (size_t i = 0; i < n_input_tensors; i++) {
      auto cur_in_shape = input_tensors_[i].shape();
      if (i != 0) {
        auto first_in_shape = input_tensors_[0].shape();
        if (cur_in_shape.size(0) != first_in_shape.size(0)) {
          HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same height");
        }
      }
      if (cur_in_shape.dims() != 2) {
        HCTR_OWN_THROW(Error_t::WrongInput, "Only 2D tensors can be concatenated");
      }
      if (i == 0) {
        height = cur_in_shape.size(0);
      }
      new_width += cur_in_shape.size(1);
    }
    core23::BufferParams buf_p{.channel = GetBlobsBufferChannel()};

    output_tensor = core23::Tensor(
        input_tensors_[0].my_params().shape({height, new_width}).buffer_params(buf_p));
    output_tensors_.push_back(output_tensor);
  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void ConcatLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  auto stream = get_gpu().get_stream();

  int n_input_tensors = input_tensors_.size();
  int block_size = 256;
  int n_blocks = get_gpu().get_sm_count() * 8;
  auto& output_tensor = output_tensors_[0];
  T* out = output_tensor.data<T>();
  const int2 out_dim = {static_cast<int>(output_tensor.shape().size(0)),
                        static_cast<int>(output_tensor.shape().size(1))};
  int offset = 0;
  for (auto& input_tensor : input_tensors_) {
    T* in = input_tensor.data<T>();
    const int2 in_dim = {static_cast<int>(input_tensor.shape().size(0)),
                         static_cast<int>(input_tensor.shape().size(1))};

    concat_fwd_kernel<<<n_blocks, block_size, 0, stream>>>(out, out_dim, in, in_dim, offset);
    offset += in_dim.y;
  }
}

template <typename T>
void ConcatLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());
  auto stream = get_gpu().get_stream();

  int block_size = 256;
  int n_blocks = get_gpu().get_sm_count() * 8;
  auto& output_tensor = output_tensors_[0];
  T* out = output_tensor.data<T>();
  const int2 out_dim = {static_cast<int>(output_tensor.shape().size(0)),
                        static_cast<int>(output_tensor.shape().size(1))};
  int grid_size = std::min(out_dim.x, n_blocks);
  int offset = 0;
  for (std::size_t i = 0; i < input_tensors_.size(); i++) {
    auto& input_tensor = input_tensors_[i];
    T* in = input_tensor.data<T>();
    const int2 in_dim = {static_cast<int>(input_tensor.shape().size(0)),
                         static_cast<int>(input_tensor.shape().size(1))};

    concat_bwd_kernel<<<grid_size, block_size, 0, stream>>>(out, out_dim, in, in_dim, offset);
    offset += in_dim.y;
  }
}

template class ConcatLayer<float>;
template class ConcatLayer<__half>;

}  // namespace HugeCTR
