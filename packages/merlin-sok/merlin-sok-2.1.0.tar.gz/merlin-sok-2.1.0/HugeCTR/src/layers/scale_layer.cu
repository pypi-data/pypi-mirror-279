#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cuda_utils.cuh>
#include <functional>
#include <include/utils.cuh>
#include <layers/scale_layer.hpp>
#include <linalg/binary_op.cuh>
#include <linalg/reduce.cuh>
#include <linalg/unary_op.cuh>
#include <network_buffer_channels.hpp>
#include <utils.hpp>

namespace HugeCTR {
namespace {

template <typename T>
void __global__ upscale_kernel(T* out, T* in, int batchsize, int num_elems, int axis, int factor) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int threads_num = blockDim.x * gridDim.x;
  int len = batchsize * num_elems;
  if (axis == 0) {
    for (int index = tid; index < len; index += threads_num) {
      for (int i = 0; i < factor; i++) {
        out[index * factor + i] = in[index];
      }
    }
  } else {
    for (int index = threadIdx.x; index < num_elems; index += blockDim.x) {
      for (int i = 0; i < factor; i++) {
        out[blockIdx.x * factor * num_elems + i * num_elems + index] =
            in[index + blockIdx.x * num_elems];
      }
    }
  }
}

template <typename T>
void __global__ downscale_kernel(T* out, T* in, int batchsize, int num_elems, int axis,
                                 int factor) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int threads_num = blockDim.x * gridDim.x;
  int len = batchsize * num_elems;
  if (axis == 0) {
    for (int index = tid; index < len; index += threads_num) {
      out[index] = in[index * factor];
    }
  } else {
    for (int index = threadIdx.x; index < num_elems; index += blockDim.x) {
      out[blockIdx.x * num_elems + index] = in[index + blockIdx.x * num_elems * factor];
    }
  }
}

template <typename T>
void scale(T* out, T* in, int batchsize, int num_elems, int axis, int factor, hipStream_t stream,
           bool forward) {
  dim3 grid(batchsize);
  dim3 block(min(num_elems, 1024));

  if (forward)
    upscale_kernel<<<grid, block, 0, stream>>>(out, in, batchsize, num_elems, axis, factor);
  else
    downscale_kernel<<<grid, block, 0, stream>>>(out, in, batchsize, num_elems, axis, factor);
}

}  // namespace

template <typename T>
ScaleLayer<T>::ScaleLayer(const core23::Tensor& in_tensor, core23::Tensor& out_tensor, int axis,
                          int factor, const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  assert(axis < 2);
  auto out_y = axis == 1 ? in_tensor.shape()[0] * factor : in_tensor.shape()[0];
  auto out_x = axis == 0 ? in_tensor.shape()[1] * factor : in_tensor.shape()[1];
  core23::Shape out_dims = {out_y, out_x};
  core23::BufferParams blobs_buffer_params = {};
  blobs_buffer_params.channel = GetBlobsBufferChannel();

  out_tensor =
      core23::Tensor(in_tensor.my_params().shape(out_dims).buffer_params(blobs_buffer_params));
  in_tensors_.push_back(in_tensor);
  out_tensors_.push_back(out_tensor);
  axis_ = axis;
  factor_ = factor;
}

template <typename T>
void ScaleLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  auto& in_tensor = in_tensors_[0];
  auto& out_tensor = out_tensors_[0];
  const auto& in_tensor_dim = in_tensor.shape();
  int axis = axis_;
  int factor = factor_;

  scale(out_tensor.data<T>(), in_tensor.data<T>(), in_tensor_dim[0], in_tensor_dim[1], axis, factor,
        get_gpu().get_stream(), true);
}

template <typename T>
void ScaleLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());
  auto& bottom_tensor = in_tensors_[0];
  auto& top_tensor = out_tensors_[0];
  const auto& bottom_tensor_dim = bottom_tensor.shape();
  int axis = axis_;
  int factor = factor_;

  scale(bottom_tensor.data<T>(), top_tensor.data<T>(), bottom_tensor_dim[0], bottom_tensor_dim[1],
        axis, factor, get_gpu().get_stream(), false);
}

template class ScaleLayer<float>;
// template class ScaleLayer<__half>;

}  // namespace HugeCTR
