#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <core23/tensor_operations.hpp>
#include <functional>
#include <layers/add_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>
namespace HugeCTR {

namespace {

template <typename T>
__global__ void add_kernel(T** inputs, T* output, int size, int num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    T tmp = 0;
    for (int i = 0; i < num; i++) {
      tmp += inputs[i][tid];
    }
    output[tid] = tmp;
  }
}

template <typename T>
__global__ void add_dgrad_kernel(const T* top_grad, T** dgrads, int size, int num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    for (int i = 0; i < num; i++) {
      dgrads[i][tid] = top_grad[tid];
    }
  }
}

template <>
__global__ void add_kernel<__half>(__half** inputs, __half* output, int size, int num) {
  const __half2** inputs2 = (const __half2**)(inputs);
  __half2* output2 = (__half2*)(output);
  int size2 = size / 2;

  const __half2 zero = __half2half2(__float2half(0.f));
  int start = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = start; i < size2; i += stride) {
    __half2 tmp = zero;
    for (int j = 0; j < num; ++j) {
      tmp += inputs2[j][i];
    }
    output2[i] = tmp;
  }
  if (start == 0 && size % 2 > 0) {
    __half tmp = __float2half(0.0f);
    for (int j = 0; j < num; ++j) {
      tmp += inputs[j][size - 1];
    }
    output[size - 1] = tmp;
  }
}

template <>
__global__ void add_dgrad_kernel<__half>(const __half* top_grad, __half** dgrads, int size,
                                         int num) {
  const __half2* top_grad2 = (const __half2*)(top_grad);
  __half2** dgrads2 = (__half2**)(dgrads);
  int size2 = size / 2;

  int start = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = start; i < size2; i += stride) {
    for (int j = 0; j < num; ++j) {
      dgrads2[j][i] = top_grad2[i];
    }
  }
  if (start == 0 && size % 2 > 0) {
    for (int j = 0; j < num; ++j) {
      dgrads[j][size - 1] = top_grad[size - 1];
    }
  }
}

}  // end of namespace

template <typename T>
AddLayer<T>::AddLayer(const std::vector<core23::Tensor>& input_tensors,
                      const core23::Tensor& output_tensor,
                      const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(input_tensors, {output_tensor}, gpu_resource), size_(input_tensors_[0].num_elements()) {
  try {
    // error input checking
    int64_t dims = input_tensors_[0].dims();
    int64_t num = input_tensors_.size();
    if (num < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "AddLayer needs at least 2 input tensors");
    }
    for (auto i = 1; i < num; i++) {
      if (input_tensors_[i].dims() != dims) {
        HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same num of dims");
      }
      for (auto j = 0; j < dims; j++) {
        if (input_tensors_[i].size(j) != input_tensors_[0].size(j)) {
          HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same dims");
        }
      }
    }

  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void AddLayer<T>::initialize() {
  CudaDeviceContext context(get_device_id());

  core23::TensorParams ptr_params =
      core23::TensorParams()
          .shape({static_cast<int64_t>(input_tensors_.size())})
          .data_type(core23::ScalarType::Pointer)
          .device({core23::DeviceType::GPU, static_cast<int8_t>(this->get_device_id())});
  input_tensor_ptr_ = core23::Tensor(ptr_params);
  std::vector<void*> ptr_cpu;
  // the in_tensors_ must be allocated before initialize() is called
  for (size_t i = 0; i < input_tensors_.size(); i++) {
    ptr_cpu.push_back(input_tensors_[i].data());
  }
  core23::copy_async(input_tensor_ptr_, ptr_cpu, get_gpu().get_stream());
}

template <typename T>
void AddLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  T* output = output_tensors_[0].data<T>();

  dim3 block_size(256, 1, 1);
  dim3 grid_size((size_ + block_size.x - 1) / block_size.x, 1, 1);
  add_kernel<<<grid_size, block_size, 0, get_gpu().get_stream()>>>(
      input_tensor_ptr_.data<T*>(), output, size_,
      static_cast<int>(input_tensor_ptr_.num_elements()));
}

template <typename T>
void AddLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());

  T* output = output_tensors_[0].data<T>();

  dim3 block_size(256, 1, 1);
  dim3 grid_size((size_ + block_size.x - 1) / block_size.x, 1, 1);
  add_dgrad_kernel<<<grid_size, block_size, 0, get_gpu().get_stream()>>>(
      output, input_tensor_ptr_.data<T*>(), size_,
      static_cast<int>(input_tensor_ptr_.num_elements()));
}

template class AddLayer<float>;
template class AddLayer<__half>;

}  // namespace HugeCTR
