#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <core23/logger.hpp>
#include <embeddings/sparse_embedding_functors.hpp>
#include <io/filesystem.hpp>
#include <utils.hpp>

namespace HugeCTR {

template <typename TypeEmbeddingComp>
std::vector<Tensors2<TypeEmbeddingComp>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<TypeEmbeddingComp>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count) {
  std::vector<Tensors2<TypeEmbeddingComp>> opt_states;
  opt_states.resize(local_gpu_count);

  for (size_t i = 0; i < local_gpu_count; ++i) {
    switch (optimizer_type) {
      case Optimizer_t::Ftrl:
        opt_states[i].push_back(opt_tensors_[i].opt_n_tensors_);
        opt_states[i].push_back(opt_tensors_[i].opt_z_tensors_);
        break;

      case Optimizer_t::Adam:
        opt_states[i].push_back(opt_tensors_[i].opt_m_tensors_);
        opt_states[i].push_back(opt_tensors_[i].opt_v_tensors_);
        break;

      case Optimizer_t::AdaGrad:
        opt_states[i].push_back(opt_tensors_[i].opt_accm_tensors_);
        break;

      case Optimizer_t::MomentumSGD:
        opt_states[i].push_back(opt_tensors_[i].opt_momentum_tensors_);
        break;

      case Optimizer_t::Nesterov:
        opt_states[i].push_back(opt_tensors_[i].opt_accm_tensors_);
        break;

      case Optimizer_t::SGD:
        break;

      default:
        throw std::runtime_error("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n");
    }
  }

  std::vector<Tensors2<TypeEmbeddingComp>> transpose_opt_states;
  if (opt_states[0].size() > 0) {
    transpose_opt_states.resize(opt_states[0].size());
    for (size_t i = 0; i < opt_states[0].size(); ++i) {
      transpose_opt_states[i].resize(opt_states.size());
      for (size_t j = 0; j < opt_states.size(); ++j) {
        transpose_opt_states[i][j] = opt_states[j][i];
      }
    }
  }
  return transpose_opt_states;
}

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::dump_opt_states(
    std::string& write_path, const ResourceManager& resource_manager,
    std::vector<Tensors2<TypeEmbeddingComp>>& opt_states) {
  size_t local_gpu_count = resource_manager.get_local_gpu_count();

  bool append_flag = false;
  CudaDeviceContext context;
  for (auto& opt_state : opt_states) {
    size_t total_size = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      total_size += opt_state[id].get_size_in_bytes();
    }
    size_t max_size = total_size;

#ifdef ENABLE_MPI
    bool is_master_process = resource_manager.is_master_process();
    HCTR_MPI_THROW(MPI_Reduce(is_master_process ? MPI_IN_PLACE : &max_size, &max_size,
                              sizeof(size_t), MPI_CHAR, MPI_MAX,
                              resource_manager.get_master_process_id(), MPI_COMM_WORLD));
#endif

    std::unique_ptr<char[]> h_opt_state(new char[max_size]);
    size_t offset = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      size_t local_size = opt_state[id].get_size_in_bytes();
      auto& local_gpu = resource_manager.get_local_gpu(id);
      context.set_device(local_gpu->get_device_id());
      HCTR_LIB_THROW(hipMemcpyAsync(h_opt_state.get() + offset, opt_state[id].get_ptr(),
                                     local_size, hipMemcpyDeviceToHost, local_gpu->get_stream()));
      offset += local_size;
    }
    sync_all_gpus(resource_manager);
    int pid = resource_manager.get_process_id();
    if (resource_manager.is_master_process()) {
      HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Write optimzer state to file" << std::endl;
      auto fs = FileSystemBuilder::build_unique_by_path(write_path);
      if (!append_flag) {
        fs->write(write_path, h_opt_state.get(), total_size, true);
        append_flag = true;
      } else {
        fs->write(write_path, h_opt_state.get(), total_size, false);
      }
    }
#ifdef ENABLE_MPI
    else {
      HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Send optimzer state to master node"
                              << std::endl;
      int tag = (pid << 8) | 0xBA;
      HCTR_MPI_THROW(MPI_Send(h_opt_state.get(), total_size, MPI_CHAR,
                              resource_manager.get_master_process_id(), tag, MPI_COMM_WORLD));
    }

    if (resource_manager.is_master_process()) {
      for (int r = 1; r < resource_manager.get_num_process(); r++) {
        HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Recv optimzer state from rank" << r
                                << ", and write to file" << std::endl;
        int tag = (r << 8) | 0xBA;
        int recv_size = 0;
        MPI_Status status;
        HCTR_MPI_THROW(MPI_Probe(r, tag, MPI_COMM_WORLD, &status));
        HCTR_MPI_THROW(MPI_Get_count(&status, MPI_CHAR, &recv_size));
        HCTR_MPI_THROW(MPI_Recv(h_opt_state.get(), recv_size, MPI_CHAR, r, tag, MPI_COMM_WORLD,
                                MPI_STATUS_IGNORE));

        auto fs = FileSystemBuilder::build_unique_by_path(write_path);
        if (!append_flag) {
          fs->write(write_path, h_opt_state.get(), recv_size, true);
          append_flag = true;
        } else {
          fs->write(write_path, h_opt_state.get(), recv_size, false);
        }
      }
    }
#endif
    HCTR_LOG(INFO, ROOT, "Done\n");
  }
}

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::load_opt_states(
    std::string& read_path, const ResourceManager& resource_manager,
    std::vector<Tensors2<TypeEmbeddingComp>>& opt_states) {
  size_t local_gpu_count = resource_manager.get_local_gpu_count();

  CudaDeviceContext context;
  size_t fs_cursor = 0;
  for (auto& opt_state : opt_states) {
    size_t total_size = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      total_size += opt_state[id].get_size_in_bytes();
    }
    int pid = resource_manager.get_process_id();

    auto h2d_op = [&opt_state, &resource_manager, &context](char* h_opt_state) {
      size_t offset = 0;
      for (size_t id = 0; id < resource_manager.get_local_gpu_count(); id++) {
        size_t local_size = opt_state[id].get_size_in_bytes();
        auto& local_gpu = resource_manager.get_local_gpu(id);
        context.set_device(local_gpu->get_device_id());
        HCTR_LIB_THROW(hipMemcpyAsync(opt_state[id].get_ptr(), h_opt_state + offset, local_size,
                                       hipMemcpyHostToDevice, local_gpu->get_stream()));
        offset += local_size;
      }
    };

    std::unique_ptr<size_t[]> proc_sizes(new size_t[resource_manager.get_num_process()]);
    proc_sizes[0] = total_size;
#ifdef ENABLE_MPI
    HCTR_MPI_THROW(MPI_Gather(&total_size, sizeof(size_t), MPI_CHAR, proc_sizes.get(),
                              sizeof(size_t), MPI_CHAR, 0, MPI_COMM_WORLD));
#endif

    if (resource_manager.is_master_process()) {
      size_t sum_sizes = 0;
      size_t max_size = 0;
      for (int i = 0; i < resource_manager.get_num_process(); ++i) {
        sum_sizes += proc_sizes[i];
        if (proc_sizes[i] > max_size) {
          max_size = proc_sizes[i];
        }
      }
      std::unique_ptr<char[]> h_opt_state(new char[max_size]);
      HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Read optimzer state from file" << std::endl;

      auto fs = FileSystemBuilder::build_unique_by_path(read_path);
      fs->read(read_path, h_opt_state.get(), total_size, fs_cursor);
      fs_cursor += total_size;

      h2d_op(h_opt_state.get());
      sync_all_gpus(resource_manager);

#ifdef ENABLE_MPI
      for (int r = 1; r < resource_manager.get_num_process(); r++) {
        HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Read from file"
                                << ", and send optimzer state to rank" << r << std::endl;
        fs->read(read_path, h_opt_state.get(), proc_sizes[r], fs_cursor);
        fs_cursor += proc_sizes[r];
        int tag = (r << 8) | 0xAB;
        HCTR_MPI_THROW(
            MPI_Send(h_opt_state.get(), proc_sizes[r], MPI_CHAR, r, tag, MPI_COMM_WORLD));
      }
#endif
    }
#ifdef ENABLE_MPI
    else {
      HCTR_LOG_S(INFO, WORLD) << "Rank" << pid
                              << ": Recv optimzer state from master node, and write to GPUs"
                              << std::endl;
      int mid = resource_manager.get_master_process_id();
      int tag = (pid << 8) | 0xAB;
      int recv_size = 0;
      MPI_Status status;
      HCTR_MPI_THROW(MPI_Probe(mid, tag, MPI_COMM_WORLD, &status));
      HCTR_MPI_THROW(MPI_Get_count(&status, MPI_CHAR, &recv_size));
      std::unique_ptr<char[]> h_opt_state(new char[recv_size]);
      auto fs = FileSystemBuilder::build_unique_by_path(read_path);
      fs->read(read_path, h_opt_state.get(), recv_size, fs_cursor);
      HCTR_MPI_THROW(MPI_Recv(h_opt_state.get(), recv_size, MPI_CHAR, mid, tag, MPI_COMM_WORLD,
                              MPI_STATUS_IGNORE));
      h2d_op(h_opt_state.get());
      sync_all_gpus(resource_manager);
    }
#endif
    HCTR_LOG(INFO, ROOT, "Done\n");
  }
}
template std::vector<Tensors2<float>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<float>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count);

template std::vector<Tensors2<__half>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<__half>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count);

template void SparseEmbeddingFunctors::dump_opt_states<float>(
    std::string& write_path, const ResourceManager& resource_manager,
    std::vector<Tensors2<float>>& opt_states);

template void SparseEmbeddingFunctors::dump_opt_states<__half>(
    std::string& write_path, const ResourceManager& resource_manager,
    std::vector<Tensors2<__half>>& opt_states);

template void SparseEmbeddingFunctors::load_opt_states<float>(
    std::string& read_path, const ResourceManager& resource_manager,
    std::vector<Tensors2<float>>& opt_states);

template void SparseEmbeddingFunctors::load_opt_states<__half>(
    std::string& read_path, const ResourceManager& resource_manager,
    std::vector<Tensors2<__half>>& opt_states);

}  // namespace HugeCTR
