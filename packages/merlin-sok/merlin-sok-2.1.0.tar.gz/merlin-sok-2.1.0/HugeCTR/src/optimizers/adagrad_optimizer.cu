#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <general_buffer2.hpp>
#include <optimizers/adagrad_optimizer.hpp>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {
namespace {

template <typename T>
__global__ void ada_grad_update4_kernel(size_t len, float* weight, const T* wgrad, float* sum,
                                        float lr, const float epsilon, float scaler) {
  size_t num_threads_in_grid = static_cast<size_t>(gridDim.x) * blockDim.x;
  constexpr int group_size = 4;
  using T4 = typename std::conditional<(sizeof(T) == 4), float4, float2>::type;
  size_t new_len = len / group_size;

  for (size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x; i < new_len;
       i += num_threads_in_grid) {
    T4 gi_group = *reinterpret_cast<const T4*>(wgrad + i * group_size);
    float gi[group_size];
#pragma unroll group_size
    for (int j = 0; j < group_size; j++) {
      gi[j] = TypeConvertFunc<float, T>::convert(reinterpret_cast<T*>(&gi_group)[j]) / scaler;
    }

    float4 accum_group = *reinterpret_cast<float4*>(sum + i * group_size);
    float* accum_ = reinterpret_cast<float*>(&accum_group);
#pragma unroll group_size
    for (int j = 0; j < group_size; j++) {
      accum_[j] += gi[j] * gi[j];
    }

    float std_[group_size];
#pragma unroll group_size
    for (int j = 0; j < group_size; j++) {
      std_[j] = epsilon + sqrtf(accum_[j]);
    }

    float4 weight_group = *reinterpret_cast<float4*>(weight + i * group_size);
    float* weight_ = reinterpret_cast<float*>(&weight_group);
#pragma unroll group_size
    for (int j = 0; j < group_size; j++) {
      weight_[j] -= lr * gi[j] / std_[j];
    }

    *reinterpret_cast<float4*>(weight + i * group_size) = weight_group;
    *reinterpret_cast<float4*>(sum + i * group_size) = accum_group;
  }

  size_t i = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x + new_len * group_size;
  if (i < len) {
    float gi = TypeConvertFunc<float, T>::convert(wgrad[i]) / scaler;
    float accum_ = sum[i];
    accum_ += gi * gi;
    float std_ = epsilon + sqrtf(accum_);
    weight[i] -= lr * gi / std_;
    sum[i] = accum_;
  }
}

template <typename T>
__global__ void ada_grad_update_kernel(int len, float* weight, const T* wgrad, float* sum, float lr,
                                       const float epsilon, float scaler) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    float gi = TypeConvertFunc<float, T>::convert(wgrad[i]) / scaler;
    float accum_ = sum[i];
    accum_ += gi * gi;
    float std_ = epsilon + sqrtf(accum_);
    weight[i] -= lr * gi / std_;
    sum[i] = accum_;
  }
}
}  // namespace

template <typename T>
AdaGradOptimizer<T>::AdaGradOptimizer(std::optional<WeightTensors> weight_tensors,
                                      std::optional<WgradTensors<T>> wgrad_tensors,
                                      const std::shared_ptr<GPUResource>& gpu_resource,
                                      float learning_rate, float initial_accu_value, float epsilon,
                                      float scaler)
    : Optimizer(weight_tensors, gpu_resource, learning_rate, scaler),
      wgrad_tensors_(wgrad_tensors),
      initial_accumulator_value_(initial_accu_value),
      epsilon_(epsilon) {
  core23::TensorParams tensor_params =
      core23::TensorParams()
          .device(core23::Device(core23::DeviceType::GPU, gpu_resource->get_device_id()))
          .data_type(core23::ScalarType::Float)
          .shape(core23::Shape({weight_tensors_->flatten().size(0)}))
          .buffer_channel(GetOptStateBufferChannnel());

  accum_tensor_ = core23::Tensor(tensor_params);
}

template <typename T>
void AdaGradOptimizer<T>::initialize() {
  HCTR_LIB_THROW(hipMemsetAsync(accum_tensor_.data(), initial_accumulator_value_,
                                 accum_tensor_.num_bytes(), gpu_resource_->get_stream()));
}

template <typename T>
void AdaGradOptimizer<T>::update() {
  CudaDeviceContext context(get_device_id());

  constexpr size_t block_dim = 256;

  auto flat_weight_tensor = weight_tensors_->flatten();
  auto flat_wgrad_tensor = wgrad_tensors_->flatten();
  float* weight = flat_weight_tensor.data();
  const T* wgrad = flat_wgrad_tensor.data();
  auto len = flat_weight_tensor.size(0);
  float* accum = accum_tensor_.data<float>();
  if (size_t(weight) % 16 == 0 && size_t(wgrad) % 16 == 0 && size_t(accum) % 16 == 0) {
    auto num_sms = gpu_resource_->get_sm_count();
    auto max_thread_per_sm = gpu_resource_->get_max_thread_per_sm();
    size_t grid_dim = num_sms * max_thread_per_sm / block_dim;
    ada_grad_update4_kernel<<<grid_dim, block_dim, 0, gpu_resource_->get_stream()>>>(
        len, weight, wgrad, accum, lr_, epsilon_, scaler_);
  } else {
    size_t grid_dim = (len - 1) / block_dim + 1;
    ada_grad_update_kernel<<<grid_dim, block_dim, 0, gpu_resource_->get_stream()>>>(
        len, weight, wgrad, accum, lr_, epsilon_, scaler_);
  }
}

template class AdaGradOptimizer<float>;
template class AdaGradOptimizer<__half>;
}  // namespace HugeCTR
