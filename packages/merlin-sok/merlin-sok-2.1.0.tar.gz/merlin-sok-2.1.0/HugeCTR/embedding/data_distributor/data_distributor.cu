/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <HugeCTR/include/utils.cuh>
#include <embedding/data_distributor/data_compression_operators.cuh>
#include <embedding/data_distributor/data_distributor.hpp>
#include <embedding/operators/communication.hpp>
#include <unordered_set>

namespace HugeCTR {

DataDistributor::DataDistributor(
    std::vector<std::shared_ptr<core::CoreResourceManager>>& core_resource_managers,
    const embedding::EmbeddingCollectionParam& ebc_param,
    const std::vector<embedding::EmbeddingTableParam>& emb_table_param_list,
    const std::vector<int>& dr_lookup_ids)
    : core_resource_managers_(core_resource_managers),
      batch_size_(ebc_param.universal_batch_size),
      batch_size_per_gpu_(ebc_param.universal_batch_size /
                          core_resource_managers[0]->get_global_gpu_count()),
      ebc_param_(ebc_param),
      emb_table_param_list_(emb_table_param_list),
      num_local_gpus_(core_resource_managers[0]->get_local_gpu_count()),
      num_global_gpus_(core_resource_managers[0]->get_global_gpu_count()),
      num_features_(ebc_param.num_lookup) {
  resident_feature_tables_ = ebc_param.shard_matrix;

  // construct lookup mappings
  for (int lookup_id = 0; lookup_id < ebc_param.num_lookup; ++lookup_id) {
    const embedding::LookupParam& lookup_param = ebc_param.lookup_params[lookup_id];
    feature_pooling_factors_.push_back(lookup_param.max_hotness);
    feature_id_to_table_id_map_[lookup_id] = lookup_param.table_id;
    for (size_t group_id = 0; group_id < ebc_param.grouped_lookup_params.size(); ++group_id) {
      if (!ebc_param.lookup_id_in_group(group_id, lookup_id)) continue;
      feature_id_to_group_id_map_[lookup_id] = group_id;
    }
  }

  init_comm_data();
  init_filtered_all_to_all();
  init_fixed_dp_bucket_range();

  for (size_t gpu_id = 0; gpu_id < num_local_gpus_; ++gpu_id) {
    data_distribution_input_.emplace_back(core_resource_managers_[gpu_id], dr_lookup_ids,
                                          ebc_param.key_type, ebc_param.offset_type);
  }
}

void DataDistributor::init_comm_data() {
  // Get number of features in each group
  size_t num_features = 0;
  for (int lookup_id = 0; lookup_id < ebc_param_.num_lookup; ++lookup_id) {
    const auto& lookup_param = ebc_param_.lookup_params[lookup_id];
    num_features += lookup_param.max_hotness;
  }

  for (size_t i = 0; i < num_local_gpus_; ++i) {
    CudaDeviceContext context(core_resource_managers_[i]->get_device_id());
    core23::Device device(core23::DeviceType::GPU, core_resource_managers_[i]->get_device_id());
    core23::TensorParams params = core23::TensorParams().device(device);

    GpuCommData comm_data;
    comm_data.last_batch_size = 0;

    size_t num_keys = num_features * ebc_param_.universal_batch_size;

    comm_data.hotness_bucket_range =
        core23::Tensor(params.shape({static_cast<int64_t>(num_features_ + 1)})
                           .data_type(core23::ScalarType::Int32));

    std::vector<int> hotness_bucket_range(1, 0);
    std::copy(feature_pooling_factors_.begin(), feature_pooling_factors_.end(),
              back_inserter(hotness_bucket_range));
    std::inclusive_scan(hotness_bucket_range.begin() + 1, hotness_bucket_range.end(),
                        hotness_bucket_range.begin() + 1);

    core23::copy_sync(comm_data.hotness_bucket_range, hotness_bucket_range);

    gpu_comm_data_.emplace_back(comm_data);
  }
}

void DataDistributor::init_filtered_all_to_all() {
  // --- allocate operators ---
  for (size_t group_id = 0; group_id < ebc_param_.grouped_lookup_params.size(); group_id++) {
    std::vector<std::unique_ptr<IDataDistributionOp>> data_distribution_ops;

    auto embedding_group_type = ebc_param_.grouped_lookup_params[group_id].embedding_group_type;
    for (size_t i = 0; i < num_local_gpus_; ++i) {
      auto core = core_resource_managers_[i];
      CudaDeviceContext context(core->get_device_id());

      if (embedding_group_type == embedding::EmbeddingGroupType::DataParallel) {
        data_distribution_ops.push_back(std::make_unique<SparseDPDataDistributionOp>(
            core, ebc_param_, group_id, emb_table_param_list_));
      } else if (embedding_group_type == embedding::EmbeddingGroupType::SparseModelParallel) {
        data_distribution_ops.push_back(std::make_unique<SparseMPDataDistributionOp>(
            core, ebc_param_, group_id, emb_table_param_list_));
      } else if (embedding_group_type == embedding::EmbeddingGroupType::DenseModelParallel ||
                 embedding_group_type ==
                     embedding::EmbeddingGroupType::DenseModelParallelWithReduction) {
        data_distribution_ops.push_back(std::make_unique<DenseMPDataDistributionOp>(
            core, ebc_param_, group_id, emb_table_param_list_));
      } else {
        HCTR_OWN_THROW(Error_t::IllegalCall,
                       "table placement strategy and embedding type not compatible");
      }
    }
    data_distribution_ops_.push_back(std::move(data_distribution_ops));
  }
}

void DataDistributor::init_fixed_dp_bucket_range() {
  // ---- init static bucket range ----
  // TODO: remove when data reader returns bucket range
  fixed_dp_bucket_range_.resize(num_local_gpus_);

  for (size_t gpu_id = 0; gpu_id < num_local_gpus_; ++gpu_id) {
    auto core = core_resource_managers_[gpu_id];
    core23::Device device(core23::DeviceType::GPU, core->get_device_id());
    core23::BufferParams buffer_params;
    buffer_params.unitary = false;
    core23::TensorParams params =
        core23::TensorParams().device(device).buffer_params(buffer_params);

    for (int lookup_id = 0; lookup_id < ebc_param_.num_lookup; ++lookup_id) {
      int num_buckets = batch_size_per_gpu_ + 1;

      auto bucket_range = core23::Tensor(
          params.shape({static_cast<int64_t>(num_buckets)}).data_type(ebc_param_.offset_type));

      fixed_dp_bucket_range_[gpu_id].push_back(std::move(bucket_range));
    }
  }

  // -- non-group specific operators
  for (size_t gpu_id = 0; gpu_id < num_local_gpus_; ++gpu_id) {
    compute_dp_bucket_range_operators_.emplace_back(core_resource_managers_[gpu_id], ebc_param_);
  }
}

void DataDistributor::distribute(int gpu_id, const std::vector<core23::Tensor>& dp_keys,
                                 const std::vector<core23::Tensor>& dp_bucket_range,
                                 DataDistributor::Result& output, int batch_size) {
  auto core = core_resource_managers_[gpu_id];
  CudaDeviceContext ctx(core->get_device_id());
  hipStream_t stream = core->get_local_gpu()->get_stream();

  const bool bucket_ranges_outdated = batch_size != gpu_comm_data_[gpu_id].last_batch_size;
  gpu_comm_data_[gpu_id].last_batch_size = batch_size;

  // sparse_forward new full batch bucket range (to be deprecated)
  // sparse_forward dp bucket ranges (to be moved to data reader)
  if (bucket_ranges_outdated) {
    compute_dp_bucket_range_operators_[gpu_id](fixed_dp_bucket_range_[gpu_id],
                                               output[0].num_keys_per_bucket, batch_size, stream);

    // Instead of recomputing for each group, copy computed result
    for (size_t grouped_id = 1; grouped_id < ebc_param_.grouped_lookup_params.size();
         ++grouped_id) {
      HCTR_LIB_THROW(hipMemcpyAsync(
          output[grouped_id].num_keys_per_bucket.data(), output[0].num_keys_per_bucket.data(),
          output[0].num_keys_per_bucket.num_bytes(), hipMemcpyDeviceToDevice, stream));
    }
  }

  data_distribution_input_[gpu_id].copy_tensor_vec(dp_keys, fixed_dp_bucket_range_[gpu_id], stream);

  for (size_t grouped_id = 0; grouped_id < ebc_param_.grouped_lookup_params.size(); grouped_id++) {
    data_distribution_ops_[grouped_id][gpu_id]->distribute(data_distribution_input_[gpu_id],
                                                           output[grouped_id], batch_size, stream);
  }
}

DataDistributor::Result allocate_output_for_data_distributor(
    std::shared_ptr<core::CoreResourceManager>& core_resource_manager,
    const embedding::EmbeddingCollectionParam& ebc_param) {
  CudaDeviceContext context(core_resource_manager->get_device_id());
  int num_global_gpus = core_resource_manager->get_global_gpu_count();
  int batch_size = ebc_param.universal_batch_size;
  int batch_size_per_gpu = ebc_param.universal_batch_size / num_global_gpus;

  DataDistributor::Result output;
  for (size_t group_id = 0; group_id < ebc_param.grouped_lookup_params.size(); ++group_id) {
    auto& grouped_lookup_params = ebc_param.grouped_lookup_params[group_id];

    int batch_size_after_filter =
        grouped_lookup_params.embedding_group_type == embedding::EmbeddingGroupType::DataParallel
            ? batch_size_per_gpu
            : batch_size;
    size_t num_buckets = 0ul;
    size_t num_features = 0ul;
    for (int lookup_id = 0; lookup_id < ebc_param.num_lookup; ++lookup_id) {
      if (!ebc_param.has_table_shard(core_resource_manager->get_global_gpu_id(), group_id,
                                     lookup_id)) {
        continue;
      }
      const auto& lookup_param = ebc_param.lookup_params[lookup_id];
      num_features += lookup_param.max_hotness;
      num_buckets += 1;
    }

    core23::Device device(core23::DeviceType::GPU, core_resource_manager->get_device_id());
    core23::TensorParams params = core23::TensorParams().device(device);

    embedding::EmbeddingInput embedding_input;
    embedding_input.h_num_keys = 0ul;
    embedding_input.keys =
        core23::Tensor(params.shape({static_cast<int64_t>(batch_size_after_filter * num_features)})
                           .data_type(ebc_param.key_type));

    embedding_input.num_keys = core23::Tensor(
        params.shape({1}).data_type(core23::ScalarType::UInt64).device(core23::DeviceType::CPU));

    embedding_input.num_keys_per_bucket = core23::Tensor(
        params.shape({static_cast<int64_t>(batch_size_per_gpu * ebc_param.num_lookup)})
            .data_type(ebc_param.offset_type));

    if (grouped_lookup_params.embedding_group_type == embedding::EmbeddingGroupType::DataParallel ||
        grouped_lookup_params.embedding_group_type ==
            embedding::EmbeddingGroupType::SparseModelParallel) {
      embedding_input.bucket_range = core23::Tensor(
          params.shape({static_cast<int64_t>(batch_size_after_filter * num_buckets + 1)})
              .data_type(ebc_param.offset_type));
    } else if (grouped_lookup_params.embedding_group_type ==
                   embedding::EmbeddingGroupType::DenseModelParallel ||
               grouped_lookup_params.embedding_group_type ==
                   embedding::EmbeddingGroupType::DenseModelParallelWithReduction) {
      auto& dense_compression_input = embedding_input.dense_compression_input;
      embedding::WgradAttr wgrad_attr;
      wgrad_attr.init(core_resource_manager, ebc_param, group_id);
      dense_compression_input.num_keys_per_table_offset =
          core23::Tensor(params.shape({static_cast<int64_t>(wgrad_attr.num_table + 1)})
                             .data_type(ebc_param.offset_type));
      dense_compression_input.table_ids =
          core23::Tensor(params.shape({static_cast<int64_t>(wgrad_attr.num_table)})
                             .data_type(core23::ScalarType::Int32));

      auto& model_parallel_compression_input =
          dense_compression_input.model_parallel_compression_input;
      model_parallel_compression_input.h_send_k_per_gpu =
          core23::Tensor(params.shape({static_cast<int64_t>(num_global_gpus)})
                             .data_type(ebc_param.offset_type)
                             .device(core23::DeviceType::CPU));
      model_parallel_compression_input.h_recv_k_per_gpu =
          core23::Tensor(params.shape({static_cast<int64_t>(num_global_gpus)})
                             .data_type(ebc_param.offset_type)
                             .device(core23::DeviceType::CPU));

      model_parallel_compression_input.model_reverse_idx =
          core23::Tensor(params.shape({static_cast<int64_t>(batch_size * num_features)})
                             .data_type(ebc_param.offset_type));
      model_parallel_compression_input.num_model_reverse_idx = 0ul;

      size_t num_features_this_group_have = 0ul;
      for (int peer_gpu_id = 0; peer_gpu_id < num_global_gpus; ++peer_gpu_id) {
        for (int lookup_id = 0; lookup_id < ebc_param.num_lookup; ++lookup_id) {
          if (!ebc_param.has_table_shard(peer_gpu_id, group_id, lookup_id)) {
            continue;
          }
          const auto& lookup_param = ebc_param.lookup_params[lookup_id];
          num_features_this_group_have += lookup_param.max_hotness;
        }
      }

      model_parallel_compression_input.network_reverse_idx = core23::Tensor(
          params.shape({static_cast<int64_t>(batch_size_per_gpu * num_features_this_group_have)})
              .data_type(ebc_param.offset_type));
      model_parallel_compression_input.num_network_reverse_idx = 0ul;
      model_parallel_compression_input.network_dst_bucket_ids = core23::Tensor(
          params.shape({static_cast<int64_t>(batch_size_per_gpu * num_features_this_group_have)})
              .data_type(ebc_param.offset_type));

      // initialize table_ids
      core23::copy_sync(dense_compression_input.table_ids, wgrad_attr.sorted_unique_table_ids);
    } else {
      HCTR_OWN_THROW(Error_t::IllegalCall, "not supported embedding_group_type.");
    }
    output.push_back(embedding_input);
  }
  return output;
}
}  // namespace HugeCTR
