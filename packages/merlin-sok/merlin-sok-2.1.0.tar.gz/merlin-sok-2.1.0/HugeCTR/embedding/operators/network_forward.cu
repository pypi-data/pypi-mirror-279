#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <embedding/operators/generic_lookup.cuh>
#include <embedding/operators/network_forward.hpp>
#include <utils.hpp>

namespace embedding {

void NetworkIndices::init(std::shared_ptr<CoreResourceManager> core,
                          const std::vector<std::vector<int>>& h_global_lookup_ids) {
  int num_gpus = static_cast<int>(h_global_lookup_ids.size());
  h_network_ids.clear();
  h_network_gpu_ids.clear();
  h_network_offsets.clear();
  h_network_dst_lookup_ids.clear();

  std::vector<std::tuple<int, int, int>> h_network_buffer_meta_info;
  for (int ggpu_id = 0; ggpu_id < num_gpus; ++ggpu_id) {
    int network_id = 0;
    for (int lookup_id : h_global_lookup_ids[ggpu_id]) {
      h_network_buffer_meta_info.push_back({ggpu_id, network_id, lookup_id});
      network_id += 1;
    }
  }

  std::sort(h_network_buffer_meta_info.begin(), h_network_buffer_meta_info.end(),
            [](const auto& lhs, const auto& rhs) { return std::get<2>(lhs) < std::get<2>(rhs); });

  for (size_t i = 0; i < h_network_buffer_meta_info.size(); ++i) {
    const auto& meta_info = h_network_buffer_meta_info[i];
    int network_gpu_id = std::get<0>(meta_info);
    int network_id = std::get<1>(meta_info);
    h_network_ids.push_back(network_id);
    h_network_gpu_ids.push_back(network_gpu_id);
  }

  int network_offset = 0;
  for (size_t i = 0; i < h_network_buffer_meta_info.size(); ++i) {
    const auto& meta_info = h_network_buffer_meta_info[i];
    int lookup_id = std::get<2>(meta_info);
    if (i == 0 || lookup_id != std::get<2>(h_network_buffer_meta_info[i - 1])) {
      h_network_offsets.push_back(network_offset);
    }
    network_offset += 1;
  }
  h_network_offsets.push_back(network_offset);

  for (size_t i = 0; i < h_network_buffer_meta_info.size(); ++i) {
    const auto& meta_info = h_network_buffer_meta_info[i];
    int lookup_id = std::get<2>(meta_info);
    if (i == 0 || lookup_id != std::get<2>(h_network_buffer_meta_info[i - 1])) {
      h_network_dst_lookup_ids.push_back(lookup_id);
    }
  }

  HugeCTR::CudaDeviceContext context(core->get_device_id());
  core23::Device device(core23::DeviceType::GPU, core->get_device_id());
  core23::BufferParams buffer_params;
  buffer_params.unitary = false;
  core23::TensorParams params = core23::TensorParams().device(device).buffer_params(buffer_params);

  this->network_ids = core23::Tensor(params.shape({static_cast<int64_t>(h_network_ids.size())})
                                         .data_type(core23::ScalarType::Int32));
  core23::copy_sync(this->network_ids, h_network_ids);
  this->network_gpu_ids =
      core23::Tensor(params.shape({static_cast<int64_t>(h_network_gpu_ids.size())})
                         .data_type(core23::ScalarType::Int32));
  core23::copy_sync(this->network_gpu_ids, h_network_gpu_ids);
  this->network_offsets =
      core23::Tensor(params.shape({static_cast<int64_t>(h_network_offsets.size())})
                         .data_type(core23::ScalarType::Int32));
  core23::copy_sync(this->network_offsets, h_network_offsets);
  this->network_dst_lookup_ids =
      core23::Tensor(params.shape({static_cast<int64_t>(h_network_dst_lookup_ids.size())})
                         .data_type(core23::ScalarType::Int32));
  core23::copy_sync(this->network_dst_lookup_ids, h_network_dst_lookup_ids);
}

void NetworkBufferAttr::init(std::shared_ptr<CoreResourceManager> core,
                             const EmbeddingCollectionParam& ebc_param, size_t grouped_id,
                             const std::vector<std::vector<int>>& h_global_lookup_ids) {
  const auto& group_params = ebc_param.grouped_lookup_params[grouped_id];
  HCTR_CHECK_HINT(group_params.embedding_group_type == EmbeddingGroupType::SparseModelParallel,
                  "UniformModelParallelEmbeddingMeta must be initialized by SparseModelParallel");

  this->num_gpus = static_cast<int>(h_global_lookup_ids.size());

  std::vector<std::vector<int>> h_id_to_ev_size;
  h_id_to_ev_size.resize(num_gpus);
  for (int ggpu_id = 0; ggpu_id < num_gpus; ++ggpu_id) {
    for (int lookup_id : h_global_lookup_ids[ggpu_id]) {
      int ev_size = ebc_param.lookup_params[lookup_id].ev_size;
      h_id_to_ev_size[ggpu_id].push_back(ev_size);
    }
  }

  std::vector<std::vector<int>> h_id_ev_start_indices;
  h_id_ev_start_indices.resize(num_gpus);
  for (int ggpu_id = 0; ggpu_id < num_gpus; ++ggpu_id) {
    h_id_ev_start_indices[ggpu_id].push_back(0);
    std::partial_sum(h_id_to_ev_size[ggpu_id].begin(), h_id_to_ev_size[ggpu_id].end(),
                     std::back_inserter(h_id_ev_start_indices[ggpu_id]));
  }

  HugeCTR::CudaDeviceContext context(core->get_device_id());
  core23::Device device(core23::DeviceType::GPU, core->get_device_id());
  core23::BufferParams buffer_params;
  buffer_params.unitary = false;
  core23::TensorParams params = core23::TensorParams().device(device).buffer_params(buffer_params);

  for (int ggpu_id = 0; ggpu_id < num_gpus; ++ggpu_id) {
    this->id_to_ev_size_list.emplace_back(
        params.shape({static_cast<int64_t>(h_id_to_ev_size[ggpu_id].size())})
            .data_type(core23::ScalarType::Int32));
    this->id_to_ev_start_indices_list.emplace_back(
        params.shape({static_cast<int64_t>(h_id_ev_start_indices[ggpu_id].size())})
            .data_type(core23::ScalarType::Int32));

    core23::copy_sync(this->id_to_ev_size_list[ggpu_id], h_id_to_ev_size[ggpu_id]);
    core23::copy_sync(this->id_to_ev_start_indices_list[ggpu_id], h_id_ev_start_indices[ggpu_id]);
  }
  this->id_to_ev_size =
      core23::init_tensor_list<int32_t>(this->id_to_ev_size_list, params.device().index());
  this->id_to_ev_start_indices =
      core23::init_tensor_list<int32_t>(this->id_to_ev_start_indices_list, params.device().index());

  this->gpu_id_to_max_ev_elements.clear();
  for (int ggpu_id = 0; ggpu_id < num_gpus; ++ggpu_id) {
    this->gpu_id_to_max_ev_elements.push_back(h_id_ev_start_indices[ggpu_id].back());
  }
  this->layout = EmbeddingLayout::FeatureMajor;
  this->max_ev_size = 0;
  for (int ggpu_id = 0; ggpu_id < num_gpus; ++ggpu_id) {
    for (auto ev_size : h_id_to_ev_size[ggpu_id]) {
      this->max_ev_size = std::max(this->max_ev_size, ev_size);
    }
  }
  this->type = ebc_param.emb_type;
}

void NetworkBuffer::init(std::shared_ptr<CoreResourceManager> core, const NetworkBufferAttr& attr,
                         int batch_size) {
  this->attr = attr;
  this->data_list.clear();

  int batch_size_per_gpu = batch_size / core->get_global_gpu_count();
  HugeCTR::CudaDeviceContext context(core->get_device_id());
  core23::Device device(core23::DeviceType::GPU, core->get_device_id());
  core23::TensorParams params = core23::TensorParams().device(device);
  for (int ggpu_id = 0; ggpu_id < attr.num_gpus; ++ggpu_id) {
    this->data_list.emplace_back(
        params.shape({batch_size_per_gpu * attr.gpu_id_to_max_ev_elements[ggpu_id]})
            .data_type(attr.type));
  }
  DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(attr.type.type(), emb_t, [&] {
    this->data = core23::init_tensor_list<emb_t>(data_list, params.device().index());
  });
}
void DenseNetworkIndices::init(std::shared_ptr<CoreResourceManager> core,
                               const std::vector<int>& h_local_hotness_range_input,
                               const std::vector<int>& h_local_hotness_input,
                               const std::vector<int>& h_ev_start_indices_input,
                               const int local_lookup_num_input, const int global_ev_offset_input) {
  this->h_local_hotness_range.assign(h_local_hotness_range_input.begin(),
                                     h_local_hotness_range_input.end());
  this->h_local_hotness.assign(h_local_hotness_input.begin(), h_local_hotness_input.end());
  this->h_ev_start_indices.assign(h_ev_start_indices_input.begin(), h_ev_start_indices_input.end());
  this->local_lookup_num = local_lookup_num_input;
  this->global_ev_offset = global_ev_offset_input;

  HugeCTR::CudaDeviceContext context(core->get_device_id());
  core23::Device device(core23::DeviceType::GPU, core->get_device_id());
  core23::BufferParams buffer_params;
  buffer_params.unitary = false;
  core23::TensorParams params = core23::TensorParams().device(device).buffer_params(buffer_params);

  this->d_local_hotness_range =
      core23::Tensor(params.shape({static_cast<int64_t>(this->h_local_hotness_range.size())})
                         .data_type(core23::ScalarType::Int32));
  core23::copy_sync(this->d_local_hotness_range, this->h_local_hotness_range);

  this->d_local_hotness =
      core23::Tensor(params.shape({static_cast<int64_t>(h_local_hotness.size())})
                         .data_type(core23::ScalarType::Int32));
  core23::copy_sync(this->d_local_hotness, this->h_local_hotness);

  this->d_ev_start_indices =
      core23::Tensor(params.shape({static_cast<int64_t>(h_ev_start_indices.size())})
                         .data_type(core23::ScalarType::Int32));
  core23::copy_sync(this->d_ev_start_indices, this->h_ev_start_indices);
}

void DenseNetworkBufferAttr::init(std::shared_ptr<CoreResourceManager> core,
                                  const EmbeddingCollectionParam& ebc_param, size_t grouped_id,
                                  int max_hotness) {
  const auto& group_params = ebc_param.grouped_lookup_params[grouped_id];
  HCTR_CHECK_HINT(
      group_params.embedding_group_type == EmbeddingGroupType::DenseModelParallel ||
          group_params.embedding_group_type == EmbeddingGroupType::DenseModelParallelWithReduction,
      "DenseNetworkBufferAttr must be initialized by DenseModelParallel or "
      "DenseModelParallelWithReduction");

  this->num_lookup = group_params.lookup_ids.size();

  HCTR_CHECK_HINT(this->num_lookup > 0, "DenseNetworkBufferAttr must have lookup , but now is <=0");
  const auto& lookup_params = ebc_param.lookup_params;
  this->ev_size = lookup_params[group_params.lookup_ids[0]].ev_size;
  this->max_hotness = max_hotness;
  this->layout = EmbeddingLayout::FeatureMajor;
  this->type = ebc_param.emb_type;
}

void DenseNetworkBuffer::init(std::shared_ptr<CoreResourceManager> core,
                              const DenseNetworkBufferAttr& attr, int batch_size) {
  this->attr = attr;

  HugeCTR::CudaDeviceContext context(core->get_device_id());

  core23::Device device(core23::DeviceType::GPU, core->get_device_id());
  core23::TensorParams params = core23::TensorParams().device(device);

  double dense_unique_ratio = get_dense_unique_ratio();
  int64_t max_num_elements = static_cast<int64_t>(batch_size) * attr.max_hotness * attr.ev_size;
  int64_t num_elements =
      static_cast<int64_t>(dense_unique_ratio * static_cast<double>(max_num_elements));
  this->data = core23::Tensor(params.shape({num_elements}).data_type(attr.type));
}

NetworkForward::NetworkForward(std::shared_ptr<CoreResourceManager> core) : core_(core) {}

namespace {
// sparse
void network_forward_to_batch_major_output(const core23::Tensor& dp_num_keys_per_bucket,
                                           const NetworkBuffer& network_buffer,
                                           const NetworkIndices& network_indices,
                                           const HugeCTR::core23::KernelParams& kernel_params,
                                           EmbeddingOutput& embedding_output, int batch_size,
                                           int gpu_id, int num_gpus, hipStream_t stream) {
  int batch_size_per_gpu = batch_size / num_gpus;
  auto& network_comm_buffer = network_buffer.data;
  auto& output_buffer = embedding_output.data;
  auto& network_attr = network_buffer.attr;
  auto& output_attr = embedding_output.attr;
  int max_ev_size = output_attr.max_ev_size;
  int num_lookup = output_attr.id_to_ev_size.num_elements();

  DISPATCH_INTEGRAL_FUNCTION_CORE23(dp_num_keys_per_bucket.data_type().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(network_comm_buffer.data_type().type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(output_buffer.data_type().type(), dst_emb_t, [&] {
        const offset_t* dp_num_keys_per_bucket_ptr = dp_num_keys_per_bucket.data<offset_t>();
        const int* network_ids_ptr = network_indices.network_ids.data<int>();
        const int* network_gpu_ids_ptr = network_indices.network_gpu_ids.data<int>();
        const int* network_offsets_ptr = network_indices.network_offsets.data<int>();
        const int* network_dst_lookup_ids_ptr = network_indices.network_dst_lookup_ids.data<int>();
        const int** network_ev_sizes_ptr = (const int**)network_attr.id_to_ev_size.data();
        const int** network_ev_offsets_ptr =
            (const int**)network_attr.id_to_ev_start_indices.data();
        const emb_t** network_comm_buffer_ptr = (const emb_t**)network_comm_buffer.data();
        const int* dst_ev_start_indices_ptr = output_attr.id_to_ev_start_indices.data<int>();
        const char* dst_combiner_ptr = output_attr.id_to_combiner.data<char>();
        dst_emb_t* output_buffer_ptr = output_buffer.data<dst_emb_t>();
        int num_network_dst_lookup_ids = network_indices.network_dst_lookup_ids.num_elements();

        auto multi_to_one_desc = make_MultiToOne<emb_t, dst_emb_t>(
            num_network_dst_lookup_ids * batch_size_per_gpu,
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = i % num_network_dst_lookup_ids;
              return bid * network_offsets_ptr[num_network_dst_lookup_ids] +
                     network_offsets_ptr[lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              if (dst_combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
                int idx = batch_size_per_gpu * lookup_id + bid;
                return static_cast<int>(dp_num_keys_per_bucket_ptr[idx]);
              } else {
                return 1;
              }
            },
            [=] __device__(int i) {
              int dst_lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];
              return dst_ev_start_indices_ptr[dst_lookup_id + 1] -
                     dst_ev_start_indices_ptr[dst_lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / network_offsets_ptr[num_network_dst_lookup_ids];
              int id = i % network_offsets_ptr[num_network_dst_lookup_ids];

              int network_gpu_id = network_gpu_ids_ptr[id];
              int network_id = network_ids_ptr[id];
              int ev_offset =
                  network_ev_offsets_ptr[network_gpu_id][network_id] * batch_size_per_gpu;
              int ev_size = network_ev_sizes_ptr[network_gpu_id][network_id];

              return network_comm_buffer_ptr[network_gpu_id] + ev_offset + bid * ev_size;
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];
              int ev_offset = dst_ev_start_indices_ptr[num_lookup] * bid;

              return output_buffer_ptr + ev_offset + dst_ev_start_indices_ptr[lookup_id];
            });
        copy_multi_to_one(multi_to_one_desc, kernel_params, max_ev_size, stream);
      });
    });
  });
}
// sparse

void network_forward_to_feature_major_output(const core23::Tensor& dp_num_keys_per_bucket,
                                             const NetworkBuffer& network_buffer,
                                             const NetworkIndices& network_indices,
                                             const HugeCTR::core23::KernelParams& kernel_params,
                                             EmbeddingOutput& embedding_output, int batch_size,
                                             int gpu_id, int num_gpus, hipStream_t stream) {
  int batch_size_per_gpu = batch_size / num_gpus;
  auto& network_comm_buffer = network_buffer.data;
  auto& output_buffer = embedding_output.data;
  auto& network_attr = network_buffer.attr;
  auto& output_attr = embedding_output.attr;
  int max_ev_size = output_attr.max_ev_size;

  DISPATCH_INTEGRAL_FUNCTION_CORE23(dp_num_keys_per_bucket.data_type().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(network_comm_buffer.data_type().type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(output_buffer.data_type().type(), dst_emb_t, [&] {
        const offset_t* dp_num_keys_per_bucket_ptr = dp_num_keys_per_bucket.data<offset_t>();
        const int* network_ids_ptr = network_indices.network_ids.data<int>();
        const int* network_gpu_ids_ptr = network_indices.network_gpu_ids.data<int>();
        const int* network_offsets_ptr = network_indices.network_offsets.data<int>();
        const int* network_dst_lookup_ids_ptr = network_indices.network_dst_lookup_ids.data<int>();
        const int** network_ev_sizes_ptr = (const int**)network_attr.id_to_ev_size.data();
        const int** network_ev_offsets_ptr =
            (const int**)network_attr.id_to_ev_start_indices.data();
        const emb_t** network_comm_buffer_ptr = (const emb_t**)network_comm_buffer.data();
        const int* dst_ev_start_indices_ptr = output_attr.id_to_ev_start_indices.data<int>();
        const char* dst_combiner_ptr = output_attr.id_to_combiner.data<char>();
        dst_emb_t* output_buffer_ptr = output_buffer.data<dst_emb_t>();
        int num_network_dst_lookup_ids = network_indices.network_dst_lookup_ids.num_elements();

        auto multi_to_one_desc = make_MultiToOne<emb_t, dst_emb_t>(
            num_network_dst_lookup_ids * batch_size_per_gpu,
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = i % num_network_dst_lookup_ids;
              return bid * network_offsets_ptr[num_network_dst_lookup_ids] +
                     network_offsets_ptr[lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              if (dst_combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
                int idx = batch_size_per_gpu * lookup_id + bid;
                return static_cast<int>(dp_num_keys_per_bucket_ptr[idx]);
              } else {
                return 1;
              }
            },
            [=] __device__(int i) {
              int dst_lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];
              return dst_ev_start_indices_ptr[dst_lookup_id + 1] -
                     dst_ev_start_indices_ptr[dst_lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / network_offsets_ptr[num_network_dst_lookup_ids];
              int id = i % network_offsets_ptr[num_network_dst_lookup_ids];

              int network_gpu_id = network_gpu_ids_ptr[id];
              int network_id = network_ids_ptr[id];
              int ev_offset =
                  network_ev_offsets_ptr[network_gpu_id][network_id] * batch_size_per_gpu;
              int ev_size = network_ev_sizes_ptr[network_gpu_id][network_id];

              return network_comm_buffer_ptr[network_gpu_id] + ev_offset + bid * ev_size;
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              int ev_offset = dst_ev_start_indices_ptr[lookup_id] * batch_size_per_gpu;
              int ev_size =
                  dst_ev_start_indices_ptr[lookup_id + 1] - dst_ev_start_indices_ptr[lookup_id];
              return output_buffer_ptr + ev_offset + bid * ev_size;
            });
        copy_multi_to_one(multi_to_one_desc, kernel_params, max_ev_size, stream);
      });
    });
  });
}

template <typename src_emb_t, typename dst_emb_t, typename offset_t>
struct DenseNetworkForwardFeatureMajorOneToOneDesc {
  using SrcT = src_emb_t;
  using DstT = dst_emb_t;

  HOST_DEVICE_INLINE int num_vec() { return num_vec_; }
  HOST_DEVICE_INLINE bool need_copy(int i) { return true; }

  HOST_DEVICE_INLINE int get_vec_length(int i) { return ev_size; }
  // we need a transform to src id use num_model_revers_idx
  HOST_DEVICE_INLINE const SrcT* get_src_ptr(int i) {
    return src_ptr + reverse_id_ptr[i] * ev_size;
  }
  HOST_DEVICE_INLINE DstT* get_dst_ptr(int i) {
    int hotness_id = bucket_id_ptr[i] / batch_size_per_gpu;
    int64_t lookup_id = bs_upper_bound_sub_one(hotness_range, range_num, hotness_id);
    offset_t bucket_id = bucket_id_ptr[i];
    hotness_id = hotness_id - hotness_range[lookup_id];
    int bid = bucket_id % batch_size_per_gpu;
    return dst_ptr + batch_size_per_gpu * ev_start_indices[lookup_id] +
           bid * hotness_list[lookup_id] * ev_size + hotness_id * ev_size;
  }

  size_t num_vec_;
  int ev_size;
  int batch_size_per_gpu;
  int range_num;
  const int* hotness_range;
  const int* ev_start_indices;
  const int* hotness_list;

  const offset_t* __restrict__ reverse_id_ptr;
  const offset_t* __restrict__ bucket_id_ptr;
  const src_emb_t* __restrict__ src_ptr;
  dst_emb_t* __restrict__ dst_ptr;
};

template <typename src_emb_t, typename dst_emb_t, typename offset_t>
struct DenseNetworkForwardBatchMajorOneToOneDesc {
  using SrcT = src_emb_t;
  using DstT = dst_emb_t;
  HOST_DEVICE_INLINE int num_vec() { return num_vec_; }

  HOST_DEVICE_INLINE bool need_copy(int i) { return true; }

  HOST_DEVICE_INLINE int get_vec_length(int i) { return ev_size; }
  // we need a transform to src id use num_model_revers_idx
  HOST_DEVICE_INLINE const SrcT* get_src_ptr(int i) {
    return src_ptr + reverse_id_ptr[i] * ev_size;
  }
  HOST_DEVICE_INLINE DstT* get_dst_ptr(int i) {
    int hotness_id = bucket_id_ptr[i] / batch_size_per_gpu;
    int64_t lookup_id = bs_upper_bound_sub_one(hotness_range, range_num, hotness_id);
    offset_t bucket_id = bucket_id_ptr[i];
    hotness_id = hotness_id - hotness_range[lookup_id];
    int bid = bucket_id % batch_size_per_gpu;

    return dst_ptr + bid * global_ev_offset + ev_start_indices[lookup_id] + hotness_id * ev_size;
  }

  size_t num_vec_;
  int ev_size;
  int batch_size_per_gpu;
  int range_num;
  int global_ev_offset;

  const int* hotness_range;
  const int* ev_start_indices;

  const offset_t* __restrict__ reverse_id_ptr;
  const offset_t* __restrict__ bucket_id_ptr;
  const src_emb_t* __restrict__ src_ptr;
  dst_emb_t* __restrict__ dst_ptr;
};

// dense
void dense_network_forward_to_batch_major_output(const EmbeddingInput& embedding_input,
                                                 const DenseNetworkBuffer& network_buffer,
                                                 const DenseNetworkIndices& network_indices,
                                                 const HugeCTR::core23::KernelParams& kernel_params,
                                                 EmbeddingOutput& embedding_output, int batch_size,
                                                 int gpu_id, int num_gpus, hipStream_t stream,
                                                 bool do_reduction) {
  int batch_size_per_gpu = batch_size / num_gpus;

  int ev_size = network_buffer.attr.ev_size;
  size_t num_key = embedding_input.dense_compression_input.model_parallel_compression_input
                       .num_network_reverse_idx;

  auto& network_comm_buffer = network_buffer.data;
  auto& output_buffer = embedding_output.data;
  auto& reverse_idx =
      embedding_input.dense_compression_input.model_parallel_compression_input.network_reverse_idx;
  auto& bucket_ids = embedding_input.dense_compression_input.model_parallel_compression_input
                         .network_dst_bucket_ids;
  auto& num_network_reverse_idx = embedding_input.dense_compression_input
                                      .model_parallel_compression_input.num_network_reverse_idx;
  DISPATCH_INTEGRAL_FUNCTION_CORE23(reverse_idx.data_type().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(network_comm_buffer.data_type().type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(output_buffer.data_type().type(), dst_emb_t, [&] {
        const emb_t* network_comm_buffer_ptr = (const emb_t*)network_comm_buffer.data();
        dst_emb_t* output_buffer_ptr = output_buffer.data<dst_emb_t>();
        offset_t* reverse_idx_ptr = reverse_idx.data<offset_t>();
        offset_t* bucket_ids_ptr = bucket_ids.data<offset_t>();
        auto hotness_range_ptr = network_indices.d_local_hotness_range.data<int>();
        auto ev_start_indices_ptr = network_indices.d_ev_start_indices.data<int>();
        int range_num = network_indices.local_lookup_num + 1;
        int global_ev_offset = network_indices.global_ev_offset;
        using CopyDesc = DenseNetworkForwardBatchMajorOneToOneDesc<emb_t, dst_emb_t, offset_t>;

        CopyDesc one_to_one_desc = {num_network_reverse_idx, ev_size,
                                    batch_size_per_gpu,      range_num,
                                    global_ev_offset,        hotness_range_ptr,
                                    ev_start_indices_ptr,    reverse_idx_ptr,
                                    bucket_ids_ptr,          network_comm_buffer_ptr,
                                    output_buffer_ptr};
        if (do_reduction) {
          copy_one_to_one(one_to_one_desc, kernel_params, ev_size, stream, true);
          one_to_one_atomic(one_to_one_desc, kernel_params, ev_size, num_network_reverse_idx,
                            stream);

        } else {
          copy_one_to_one(one_to_one_desc, kernel_params, ev_size, stream, false);
        }
      });
    });
  });
}

// network is input;
// output is embedding_output
void dense_network_forward_to_feature_major_output(
    const EmbeddingInput& embedding_input, const DenseNetworkBuffer& network_buffer,
    const DenseNetworkIndices& network_indices, const HugeCTR::core23::KernelParams& kernel_params,
    EmbeddingOutput& embedding_output, int batch_size, int gpu_id, int num_gpus,
    hipStream_t stream, bool do_reduction) {
  int batch_size_per_gpu = batch_size / num_gpus;

  int ev_size = network_buffer.attr.ev_size;
  size_t num_key = embedding_input.dense_compression_input.model_parallel_compression_input
                       .num_network_reverse_idx;

  auto& network_comm_buffer = network_buffer.data;
  auto& output_buffer = embedding_output.data;
  auto& reverse_idx =
      embedding_input.dense_compression_input.model_parallel_compression_input.network_reverse_idx;
  auto& bucket_ids = embedding_input.dense_compression_input.model_parallel_compression_input
                         .network_dst_bucket_ids;
  auto& num_network_reverse_idx = embedding_input.dense_compression_input
                                      .model_parallel_compression_input.num_network_reverse_idx;
  DISPATCH_INTEGRAL_FUNCTION_CORE23(reverse_idx.data_type().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(network_comm_buffer.data_type().type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(output_buffer.data_type().type(), dst_emb_t, [&] {
        const emb_t* network_comm_buffer_ptr = (const emb_t*)network_comm_buffer.data();
        dst_emb_t* output_buffer_ptr = output_buffer.data<dst_emb_t>();
        offset_t* reverse_idx_ptr = reverse_idx.data<offset_t>();
        offset_t* bucket_ids_ptr = bucket_ids.data<offset_t>();
        int range_num = network_indices.local_lookup_num + 1;

        auto hotness_range_ptr = network_indices.d_local_hotness_range.data<int>();
        auto ev_start_indices_ptr = network_indices.d_ev_start_indices.data<int>();
        auto hotness_list = network_indices.d_local_hotness.data<int>();
        using CopyDesc = DenseNetworkForwardFeatureMajorOneToOneDesc<emb_t, dst_emb_t, offset_t>;
        CopyDesc one_to_one_desc = {num_network_reverse_idx,
                                    ev_size,
                                    batch_size_per_gpu,
                                    range_num,
                                    hotness_range_ptr,
                                    ev_start_indices_ptr,
                                    hotness_list,
                                    reverse_idx_ptr,
                                    bucket_ids_ptr,
                                    network_comm_buffer_ptr,
                                    output_buffer_ptr};

        if (do_reduction) {
          copy_one_to_one(one_to_one_desc, kernel_params, ev_size, stream, true);
          one_to_one_atomic(one_to_one_desc, kernel_params, ev_size, num_network_reverse_idx,
                            stream);
        } else {
          copy_one_to_one(one_to_one_desc, kernel_params, ev_size, stream, false);
        }
      });
    });
  });
}

}  // namespace

void NetworkForward::sparse_forward(const core23::Tensor& dp_num_keys_per_bucket,
                                    const NetworkBuffer& network_buffer,
                                    const NetworkIndices& network_indices,
                                    EmbeddingOutput& embedding_output, int batch_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();
  int gpu_id = core_->get_global_gpu_id();
  int num_gpus = core_->get_global_gpu_count();

  if (embedding_output.attr.layout == EmbeddingLayout::FeatureMajor) {
    network_forward_to_feature_major_output(dp_num_keys_per_bucket, network_buffer, network_indices,
                                            core_->get_kernel_param(), embedding_output, batch_size,
                                            gpu_id, num_gpus, stream);
  } else {
    HCTR_ASSERT(embedding_output.attr.layout == EmbeddingLayout::BatchMajor);
    network_forward_to_batch_major_output(dp_num_keys_per_bucket, network_buffer, network_indices,
                                          core_->get_kernel_param(), embedding_output, batch_size,
                                          gpu_id, num_gpus, stream);
  }
}

void NetworkForward::dense_forward(const EmbeddingInput& embedding_input,
                                   const DenseNetworkBuffer& network_buffer,
                                   const DenseNetworkIndices& network_indices,
                                   EmbeddingOutput& embedding_output, int batch_size,
                                   bool do_reduction) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();
  int gpu_id = core_->get_global_gpu_id();
  int num_gpus = core_->get_global_gpu_count();

  if (embedding_output.attr.layout == EmbeddingLayout::FeatureMajor) {
    dense_network_forward_to_feature_major_output(
        embedding_input, network_buffer, network_indices, core_->get_kernel_param(),
        embedding_output, batch_size, gpu_id, num_gpus, stream, do_reduction);
  } else {
    HCTR_ASSERT(embedding_output.attr.layout == EmbeddingLayout::BatchMajor);
    dense_network_forward_to_batch_major_output(embedding_input, network_buffer, network_indices,
                                                core_->get_kernel_param(), embedding_output,
                                                batch_size, gpu_id, num_gpus, stream, do_reduction);
  }
}

void NetworkForward::compute(
    const core23::Tensor& row_lengths, const core23::Tensor& d_combiner_list,
    const core23::Tensor& network_comm_buffer, const core23::Tensor& network_ids,
    const core23::Tensor& network_gpu_ids, const core23::Tensor& network_offsets,
    const core23::Tensor& network_dst_lookup_ids, const core23::Tensor& network_ev_sizes,
    const core23::Tensor& network_ev_offsets, core23::Tensor& output_buffer,
    const core23::Tensor& d_ev_size_offset, int batch_size, int max_ev_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  int batch_size_per_gpu = batch_size / core_->get_global_gpu_count();
  DISPATCH_INTEGRAL_FUNCTION_CORE23(row_lengths.data_type().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(network_comm_buffer.data_type().type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(output_buffer.data_type().type(), dst_emb_t, [&] {
        auto stream = core_->get_local_gpu()->get_stream();

        const offset_t** row_lengths_ptr = (const offset_t**)row_lengths.data();
        const int* network_ids_ptr = network_ids.data<int>();
        const int* network_gpu_ids_ptr = network_gpu_ids.data<int>();
        const int* network_offsets_ptr = network_offsets.data<int>();
        const int* network_dst_lookup_ids_ptr = network_dst_lookup_ids.data<int>();
        const int** network_ev_sizes_ptr = (const int**)network_ev_sizes.data();
        const int** network_ev_offsets_ptr = (const int**)network_ev_offsets.data();
        const emb_t** network_comm_buffer_ptr = (const emb_t**)network_comm_buffer.data();
        const int* d_ev_size_offset_ptr = d_ev_size_offset.data<int>();
        const char* combiner_ptr = d_combiner_list.data<char>();
        dst_emb_t** output_buffer_ptr = (dst_emb_t**)output_buffer.data();
        int num_network_dst_lookup_ids = network_dst_lookup_ids.num_elements();
        int gpu_id = core_->get_global_gpu_id();

        auto multi_to_one_desc = make_MultiToOne<emb_t, dst_emb_t>(
            num_network_dst_lookup_ids * batch_size_per_gpu,
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = i % num_network_dst_lookup_ids;
              return bid * network_offsets_ptr[num_network_dst_lookup_ids] +
                     network_offsets_ptr[lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              if (combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
                return static_cast<int>(row_lengths_ptr[lookup_id][bid]);
              } else {
                return 1;
              }
            },
            [=] __device__(int i) {
              int dst_lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];
              return d_ev_size_offset_ptr[dst_lookup_id + 1] - d_ev_size_offset_ptr[dst_lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / network_offsets_ptr[num_network_dst_lookup_ids];
              int id = i % network_offsets_ptr[num_network_dst_lookup_ids];

              int network_gpu_id = network_gpu_ids_ptr[id];
              int network_id = network_ids_ptr[id];
              int ev_offset =
                  network_ev_offsets_ptr[network_gpu_id][network_id] * batch_size_per_gpu;
              int ev_size = network_ev_sizes_ptr[network_gpu_id][network_id];

              return network_comm_buffer_ptr[network_gpu_id] + ev_offset + bid * ev_size;
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              int ev_size = d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
              return output_buffer_ptr[lookup_id] + bid * ev_size;
            });
        copy_multi_to_one(multi_to_one_desc, max_ev_size, stream);
      });
    });
  });
}

}  // namespace embedding
