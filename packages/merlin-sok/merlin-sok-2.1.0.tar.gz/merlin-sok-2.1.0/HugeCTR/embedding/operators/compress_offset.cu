#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <embedding/common.hpp>
#include <embedding/operators/compress_offset.hpp>
#include <embedding/operators/generic_lookup.cuh>
#include <utils.hpp>

namespace embedding {

template <typename offset_t>
__global__ void compress_offset_kernel(const offset_t *offset, int num, int stride,
                                       offset_t *compressed_offset) {
  CUDA_1D_KERNEL_LOOP(i, num) { compressed_offset[i] = offset[i * stride]; }
}

CompressOffset::CompressOffset(std::shared_ptr<CoreResourceManager> core, int num_compressed_offset,
                               core23::DataType type)
    : core_(core), num_compressed_offset_(num_compressed_offset) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  core23::Device device(core23::DeviceType::GPU, core->get_device_id());
  core23::TensorParams params = core23::TensorParams().device(device);

  compressed_offset_ = core23::Tensor(params.shape({num_compressed_offset}).data_type(type));
}

void CompressOffset::compute(const core23::Tensor &offset, int batch_size,
                             core23::Tensor *compressed_offset) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();
  HCTR_CHECK(offset.data_type() == compressed_offset_.data_type());

  DISPATCH_INTEGRAL_FUNCTION_CORE23(offset.data_type().type(), offset_t, [&] {
    dim3 block_size(256);

    compress_offset_kernel<<<1, block_size, 0, stream>>>(offset.data<offset_t>(),
                                                         num_compressed_offset_, batch_size,
                                                         compressed_offset_.data<offset_t>());
  });

  *compressed_offset = compressed_offset_;
}

AverageCombiner::AverageCombiner(std::shared_ptr<CoreResourceManager> core, int num_gpus,
                                 int num_local_embedding, const std::vector<int> &ev_size_list,
                                 int universal_batch_size)
    : core_(core), num_gpus_(num_gpus), num_local_embedding_(num_local_embedding) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  core23::Device device(core23::DeviceType::GPU, core->get_device_id());
  core23::TensorParams params = core23::TensorParams().device(device);

  int num_ev_size_sum = std::accumulate(ev_size_list.begin(), ev_size_list.end(), 0);

  // TODO: The float emb vec can be reduced to the scale of num of local embedding
  float_emb_vec_ = core23::Tensor(params.shape({universal_batch_size / num_gpus, num_ev_size_sum})
                                      .data_type(core23::ScalarType::Float));
}

void AverageCombiner::compute_feature_major(const core23::Tensor &dp_num_keys_per_bucket,
                                            const core23::Tensor &src_emb_vec,
                                            const core23::Tensor &d_local_embedding_list,
                                            const core23::Tensor &d_combiner_list,
                                            const core23::Tensor &d_ev_size_offset, int batch_size,
                                            int max_ev_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  int gpu_id = core_->get_global_gpu_id();
  auto stream = core_->get_local_gpu()->get_stream();
  int batch_size_per_gpu = batch_size / num_gpus_;

  DISPATCH_INTEGRAL_FUNCTION_CORE23(dp_num_keys_per_bucket.data_type().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(src_emb_vec.data_type().type(), emb_t, [&] {
      const offset_t *dp_num_keys_per_bucket_ptr = dp_num_keys_per_bucket.data<offset_t>();
      const int *local_embedding_ptr = d_local_embedding_list.data<int>();
      const int *d_ev_size_offset_ptr = d_ev_size_offset.data<int>();
      const emb_t *top_grad_ptr = src_emb_vec.data<emb_t>();
      const char *combiner_ptr = d_combiner_list.data<char>();
      float *float_emb_vec_ptr = float_emb_vec_.data<float>();
      int gpu_id = core_->get_global_gpu_id();

      auto multi_to_one_desc = make_MultiToOne<emb_t, float>(
          batch_size_per_gpu * num_local_embedding_, [=] __device__(int i) { return i; },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            if (combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
              int idx = batch_size_per_gpu * lookup_id + bid;
              return static_cast<int>(dp_num_keys_per_bucket_ptr[idx]);
            } else {
              return 1;
            }
          },
          [=] __device__(int i) {
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];
            return d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
          },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            int ev_offset = d_ev_size_offset_ptr[lookup_id] * batch_size_per_gpu;
            int ev_size = d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
            return top_grad_ptr + ev_offset + bid * ev_size;
          },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            int ev_offset = d_ev_size_offset_ptr[lookup_id] * batch_size_per_gpu;
            int ev_size = d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
            return float_emb_vec_ptr + ev_offset + bid * ev_size;
          });
      copy_multi_to_one(multi_to_one_desc, max_ev_size, stream);
    });
  });
}

void AverageCombiner::compute_batch_major(const core23::Tensor &dp_num_keys_per_bucket,
                                          const core23::Tensor &src_emb_vec,
                                          const core23::Tensor &d_local_embedding_list,
                                          const core23::Tensor &d_combiner_list,
                                          const core23::Tensor &d_ev_size_offset, int batch_size,
                                          int max_ev_size, int num_lookup) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  int gpu_id = core_->get_global_gpu_id();
  auto stream = core_->get_local_gpu()->get_stream();
  int batch_size_per_gpu = batch_size / num_gpus_;

  DISPATCH_INTEGRAL_FUNCTION_CORE23(dp_num_keys_per_bucket.data_type().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(src_emb_vec.data_type().type(), emb_t, [&] {
      const offset_t *dp_num_keys_per_bucket_ptr = dp_num_keys_per_bucket.data<offset_t>();
      const int *local_embedding_ptr = d_local_embedding_list.data<int>();
      const int *d_ev_size_offset_ptr = d_ev_size_offset.data<int>();
      const emb_t *top_grad_ptr = src_emb_vec.data<emb_t>();
      const char *combiner_ptr = d_combiner_list.data<char>();
      float *float_emb_vec_ptr = float_emb_vec_.data<float>();
      int gpu_id = core_->get_global_gpu_id();

      auto multi_to_one_desc = make_MultiToOne<emb_t, float>(
          batch_size_per_gpu * num_lookup, [=] __device__(int i) { return i; },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            if (combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
              int idx = batch_size_per_gpu * lookup_id + bid;
              return static_cast<int>(dp_num_keys_per_bucket_ptr[idx]);
            } else {
              return 1;
            }
          },
          [=] __device__(int i) {
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];
            return d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
          },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            int batch_ev_offset = d_ev_size_offset_ptr[num_lookup] * bid;
            ;
            int ev_offset = d_ev_size_offset_ptr[lookup_id];
            return top_grad_ptr + batch_ev_offset + ev_offset;
          },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            int batch_ev_offset = d_ev_size_offset_ptr[num_lookup] * bid;
            int ev_offset = d_ev_size_offset_ptr[lookup_id];
            return float_emb_vec_ptr + batch_ev_offset + ev_offset;
          });
      copy_multi_to_one(multi_to_one_desc, max_ev_size, stream);
    });
  });
}

}  // namespace embedding
