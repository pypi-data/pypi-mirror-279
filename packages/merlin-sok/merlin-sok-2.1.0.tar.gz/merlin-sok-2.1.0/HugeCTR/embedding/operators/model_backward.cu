#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <embedding/operators/generic_lookup.cuh>
#include <embedding/operators/index_calculation.hpp>
#include <embedding/operators/model_backward.hpp>
#include <embedding/operators/model_forward.hpp>
#include <embedding/operators/multi_to_one_reduce.cuh>
#include <embedding/operators/multi_to_one_reduce_v2.cuh>
#include <utils.cuh>
#include <utils.hpp>

namespace embedding {

template <typename T>
__global__ void mp_cal_src_ptrs(const T** src_ptr, const uint32_t* src_ids_ptr, int batch_size,
                                int batch_size_per_gpu, const size_t num_keys,
                                const int* src_id_to_ev_size_ptr,
                                const int* src_id_to_ev_start_indices_ptr, T** src_ptrs) {
  CUDA_1D_KERNEL_LOOP(i, num_keys) {
    uint32_t bucket_id = src_ids_ptr[i];
    int embedding_id = bucket_id / batch_size;
    int batch_id = bucket_id % batch_size;
    int gpu_id = batch_id / batch_size_per_gpu;
    int local_batch_id = batch_id % batch_size_per_gpu;
    int ev_size = src_id_to_ev_size_ptr[i];
    src_ptrs[i] = const_cast<T*>(src_ptr[gpu_id] +
                                 batch_size_per_gpu * src_id_to_ev_start_indices_ptr[embedding_id] +
                                 local_batch_id * ev_size);
  }
  return;
}

template <typename T>
__global__ void mp_cal_src_ptrs_same_ev_size(const T** src_ptr, const uint32_t* src_ids_ptr,
                                             int batch_size, int batch_size_per_gpu,
                                             const size_t num_keys, const int ev_size,
                                             const int* src_id_to_ev_start_indices_ptr,
                                             T** src_ptrs) {
  CUDA_1D_KERNEL_LOOP(i, num_keys) {
    uint32_t bucket_id = src_ids_ptr[i];
    int embedding_id = bucket_id / batch_size;
    int batch_id = bucket_id % batch_size;
    int gpu_id = batch_id / batch_size_per_gpu;
    int local_batch_id = batch_id % batch_size_per_gpu;
    src_ptrs[i] = const_cast<T*>(src_ptr[gpu_id] +
                                 batch_size_per_gpu * src_id_to_ev_start_indices_ptr[embedding_id] +
                                 local_batch_id * ev_size);
  }
  return;
}

void LocalReduce::init(std::shared_ptr<CoreResourceManager> core, int max_ev_size,
                       size_t max_input_num) {
  HugeCTR::CudaDeviceContext ctx(core->get_device_id());

  this->core_ = core;

  int num_sms = core_->get_kernel_param().num_sms;
  int max_partial_num = (max_input_num - 1) / EV_NUM + 1;
  core23::Device device(core23::DeviceType::GPU, core->get_device_id());

  this->partial_reduce_result_.partial_wgrad =
      core23::Tensor(core23::TensorParams()
                         .shape({num_sms * 4 * max_ev_size})
                         .data_type(core23::ScalarType::Float)
                         .device(device));
  this->partial_reduce_result_.partial_keys =
      core23::Tensor(core23::TensorParams()
                         .shape({num_sms * 4})
                         .data_type(core23::ScalarType::UInt32)
                         .device(device));
  this->partial_reduce_result_.partial_ev_length =
      core23::Tensor(core23::TensorParams()
                         .shape({num_sms * 4})
                         .data_type(core23::ScalarType::Int32)
                         .device(device));
  this->partial_reduce_result_.partial_dst_offset_array =
      core23::Tensor(core23::TensorParams()
                         .shape({num_sms * 4})
                         .data_type(core23::ScalarType::UInt32)
                         .device(device));
  this->partial_reduce_result_.partial_wgrad_new =
      core23::Tensor(core23::TensorParams()
                         .shape({max_partial_num * max_ev_size})
                         .data_type(core23::ScalarType::Float)
                         .device(device));
  this->partial_reduce_result_.partial_ev_length_new =
      core23::Tensor(core23::TensorParams()
                         .shape({max_partial_num})
                         .data_type(core23::ScalarType::Int32)
                         .device(device));
  this->partial_reduce_result_.partial_dst_id_array_new =
      core23::Tensor(core23::TensorParams()
                         .shape({max_partial_num})
                         .data_type(core23::ScalarType::UInt32)
                         .device(device));
  this->partial_reduce_result_.src_ptrs =
      core23::Tensor(core23::TensorParams()
                         .shape({static_cast<int64_t>(max_input_num * sizeof(char*))})
                         .data_type(core23::ScalarType::Char)
                         .device(device));

  this->partial_reduce_result_.max_input_num = max_input_num;
}

void LocalReduce::local_reduce(const ReductionIndices& reduction_indices,
                               const ModelCommBuffer& src_buffer, Wgrad& wgrad, int batch_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();

  const auto& src_buffer_attr = src_buffer.attr;
  const auto& dst_attr = wgrad.attr;
  if (src_buffer_attr.num_lookup == 0 || reduction_indices.num_elements == 0) return;

  int batch_size_per_gpu = batch_size / src_buffer_attr.num_gpus;
  HCTR_CHECK_HINT(src_buffer_attr.layout == EmbeddingLayout::FeatureMajor,
                  "local reduce model comm buffer should be feature major");

  const int* src_id_to_ev_start_indices_ptr = src_buffer_attr.id_to_ev_start_indices.data<int>();
  const int* src_id_to_ev_size_ptr = reduction_indices.ev_sizes.data<int>();
  const uint32_t* src_ids_ptr = reduction_indices.src_ids.data<uint32_t>();

  const int* dst_table_id_to_ev_size_ptr = dst_attr.table_id_to_ev_size.data<int>();

  const int* dst_table_ids_ptr = wgrad.table_ids.data<int>();
  const uint32_t* dst_ev_start_indices_ptr = wgrad.ev_start_indices.data<uint32_t>();
  const uint32_t* dst_ids_ptr = reduction_indices.dst_ids.data<uint32_t>();
  if (wgrad.attr.is_same_ev_size) {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(src_buffer.attr.type.type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(wgrad.data.data_type().type(), grad_t, [&] {
        const emb_t** src_ptr = (const emb_t**)src_buffer.data.data();
        grad_t* dst_ptr = wgrad.data.data<grad_t>();
        emb_t** src_ptrs = (emb_t**)this->partial_reduce_result_.src_ptrs.data();

        mp_cal_src_ptrs_same_ev_size<<<core_->get_kernel_param().num_sms * 8, 256, 0, stream>>>(
            src_ptr, src_ids_ptr, batch_size, batch_size_per_gpu, reduction_indices.num_elements,
            wgrad.attr.same_ev_size, src_id_to_ev_start_indices_ptr, src_ptrs);

        auto multi_to_one_desc_first_stage = make_MultiToOne_reduce_new<emb_t, grad_t>(
            [=] __device__() { return reduction_indices.num_elements; },
            [=] __device__(int i) { return src_id_to_ev_size_ptr[i]; },
            [=] __device__(int i) { return dst_ids_ptr[i]; },
            [=] __device__(int i) {
              // model buffer bucket id layout:
              // gpu i:
              //   0, ..., batch_size_per_gpu | batch_size, ..., batch_size + batch_size_per_gpu |
              //   ... | i * batch_size, ..., i * batch_size + batch_size_per_gpu | ...
              // uint32_t bucket_id = src_ids_ptr[i];
              // int embedding_id = bucket_id / batch_size;
              // int batch_id = bucket_id % batch_size;
              // int gpu_id = batch_id / batch_size_per_gpu;
              // int local_batch_id = batch_id % batch_size_per_gpu;
              // int ev_size = src_id_to_ev_size_ptr[i];
              // return src_ptr[gpu_id] +
              //       batch_size_per_gpu * src_id_to_ev_start_indices_ptr[embedding_id] +
              //       local_batch_id * ev_size;
              return src_ptrs[i];
            },
            [=] __device__(int i) {
              auto tmp_index = dst_ids_ptr[i];
              return dst_ptr + dst_ev_start_indices_ptr[tmp_index];
            });
        multi_to_one_reduce_v2(multi_to_one_desc_first_stage, reduction_indices,
                               core_->get_kernel_param(), partial_reduce_result_, wgrad,
                               src_buffer.attr.max_ev_size, stream);
      });
    });

  } else {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(src_buffer.attr.type.type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(wgrad.data.data_type().type(), grad_t, [&] {
        const emb_t** src_ptr = (const emb_t**)src_buffer.data.data();
        grad_t* dst_ptr = wgrad.data.data<grad_t>();
        emb_t** src_ptrs = (emb_t**)this->partial_reduce_result_.src_ptrs.data();

        mp_cal_src_ptrs<<<core_->get_kernel_param().num_sms * 8, 256, 0, stream>>>(
            src_ptr, src_ids_ptr, batch_size, batch_size_per_gpu, reduction_indices.num_elements,
            src_id_to_ev_size_ptr, src_id_to_ev_start_indices_ptr, src_ptrs);

        auto multi_to_one_desc_first_stage = make_MultiToOne_reduce_new<emb_t, grad_t>(
            [=] __device__() { return reduction_indices.num_elements; },
            [=] __device__(int i) { return src_id_to_ev_size_ptr[i]; },
            [=] __device__(int i) { return dst_ids_ptr[i]; },
            [=] __device__(int i) {
              // model buffer bucket id layout:
              // gpu i:
              //   0, ..., batch_size_per_gpu | batch_size, ..., batch_size + batch_size_per_gpu |
              //   ... | i * batch_size, ..., i * batch_size + batch_size_per_gpu | ...
              // uint32_t bucket_id = src_ids_ptr[i];
              // int embedding_id = bucket_id / batch_size;
              // int batch_id = bucket_id % batch_size;
              // int gpu_id = batch_id / batch_size_per_gpu;
              // int local_batch_id = batch_id % batch_size_per_gpu;
              // int ev_size = src_id_to_ev_size_ptr[i];
              // return src_ptr[gpu_id] +
              //       batch_size_per_gpu * src_id_to_ev_start_indices_ptr[embedding_id] +
              //       local_batch_id * ev_size;
              return src_ptrs[i];
            },
            [=] __device__(int i) {
              auto tmp_index = dst_ids_ptr[i];
              return dst_ptr + dst_ev_start_indices_ptr[tmp_index];
            });
        multi_to_one_reduce_v2(multi_to_one_desc_first_stage, reduction_indices,
                               core_->get_kernel_param(), partial_reduce_result_, wgrad,
                               src_buffer.attr.max_ev_size, stream);
      });
    });
  }
}

void dp_local_reduce_from_feature_major_top_grad(
    const HugeCTR::core23::KernelParams& kernel_params, const ReductionIndices& reduction_indices,
    const EmbeddingOutput& src_buffer, const core23::Tensor& local_lookup_ids, int num_lookup,
    Wgrad& wgrad, PartialReduceResult& partial_reduce_result, int batch_size_per_gpu,
    int max_ev_size, hipStream_t stream) {
  const auto& src_buffer_attr = src_buffer.attr;
  const auto& dst_attr = wgrad.attr;

  HCTR_CHECK_HINT(src_buffer_attr.layout == EmbeddingLayout::FeatureMajor,
                  "local reduce model comm buffer should be feature major");

  const int* local_lookup_ids_ptr = local_lookup_ids.data<int>();

  const int* src_id_to_ev_start_indices_ptr = src_buffer_attr.id_to_ev_start_indices.data<int>();
  const int* src_id_to_ev_size_ptr = reduction_indices.ev_sizes.data<int>();

  const uint32_t* src_ids_ptr = reduction_indices.src_ids.data<uint32_t>();

  const int* dst_table_id_to_ev_size_ptr = dst_attr.table_id_to_ev_size.data<int>();

  const int* dst_table_ids_ptr = wgrad.table_ids.data<int>();
  const uint32_t* dst_ev_start_indices_ptr = wgrad.ev_start_indices.data<uint32_t>();
  const uint32_t* dst_ids_ptr = reduction_indices.dst_ids.data<uint32_t>();

  DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(src_buffer.attr.type.type(), emb_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(wgrad.data.data_type().type(), grad_t, [&] {
      const emb_t* src_ptr = src_buffer.data.data<emb_t>();
      grad_t* dst_ptr = wgrad.data.data<grad_t>();
      auto multi_to_one_desc_first_stage = make_MultiToOne_reduce_new<emb_t, grad_t>(
          [=] __device__() { return reduction_indices.num_elements; },
          [=] __device__(int i) { return src_id_to_ev_size_ptr[i]; },
          [=] __device__(int i) { return dst_ids_ptr[i]; },

          [=] __device__(int i) {
            // top grad buffer bucket id layout:
            // 0, ..., batch_size_per_gpu - 1 | batch_size_per_gpu, ..., 2 * batch_size_per_gpu - 1|
            // ...
            uint32_t bucket_id = src_ids_ptr[i];
            int local_lookup_id = bucket_id / batch_size_per_gpu;
            int lookup_id = local_lookup_ids_ptr[local_lookup_id];

            int batch_id = bucket_id % batch_size_per_gpu;
            int ev_size = src_id_to_ev_size_ptr[i];
            return src_ptr + batch_size_per_gpu * src_id_to_ev_start_indices_ptr[lookup_id] +
                   batch_id * ev_size;
          },
          [=] __device__(int i) {
            auto tmp_index = dst_ids_ptr[i];
            return dst_ptr + dst_ev_start_indices_ptr[tmp_index];
          });

      multi_to_one_reduce_v2(multi_to_one_desc_first_stage, reduction_indices, kernel_params,
                             partial_reduce_result, wgrad, src_buffer.attr.max_ev_size, stream);
    });
  });
}

void dp_local_reduce_from_batch_major_top_grad(
    const HugeCTR::core23::KernelParams& kernel_params, const ReductionIndices& reduction_indices,
    const EmbeddingOutput& src_buffer, const core23::Tensor& local_lookup_ids, int num_lookup,
    Wgrad& wgrad, PartialReduceResult& partial_reduce_result, int batch_size_per_gpu,
    int max_ev_size, hipStream_t stream) {
  const auto& src_buffer_attr = src_buffer.attr;
  const auto& dst_attr = wgrad.attr;

  HCTR_CHECK_HINT(src_buffer_attr.layout == EmbeddingLayout::BatchMajor,
                  "local reduce model comm buffer should be batch major");

  const int* local_lookup_ids_ptr = local_lookup_ids.data<int>();

  const int* src_id_to_ev_start_indices_ptr = src_buffer_attr.id_to_ev_start_indices.data<int>();
  const int* src_id_to_ev_size_ptr = reduction_indices.ev_sizes.data<int>();

  const uint32_t* src_ids_ptr = reduction_indices.src_ids.data<uint32_t>();

  const int* dst_table_id_to_ev_size_ptr = dst_attr.table_id_to_ev_size.data<int>();

  const int* dst_table_ids_ptr = wgrad.table_ids.data<int>();
  const uint32_t* dst_ev_start_indices_ptr = wgrad.ev_start_indices.data<uint32_t>();
  const uint32_t* dst_ids_ptr = reduction_indices.dst_ids.data<uint32_t>();

  DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(src_buffer.attr.type.type(), emb_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(wgrad.data.data_type().type(), grad_t, [&] {
      const emb_t* src_ptr = src_buffer.data.data<emb_t>();

      grad_t* dst_ptr = wgrad.data.data<grad_t>();
      auto multi_to_one_desc_first_stage = make_MultiToOne_reduce_new<emb_t, grad_t>(
          [=] __device__() { return reduction_indices.num_elements; },
          [=] __device__(int i) { return src_id_to_ev_size_ptr[i]; },
          [=] __device__(int i) { return dst_ids_ptr[i]; },
          [=] __device__(int i) {
            // top grad buffer bucket id layout:
            // 0, ..., num_lookup - 1 | num_lookup, ... | ... batch_size_per_gpu * num_lookup - 1
            uint32_t bucket_id = src_ids_ptr[i];
            int batch_id = bucket_id % batch_size_per_gpu;
            int lookup_id = local_lookup_ids_ptr[bucket_id / batch_size_per_gpu];

            return src_ptr + batch_id * src_id_to_ev_start_indices_ptr[num_lookup] +
                   src_id_to_ev_start_indices_ptr[lookup_id];
          },
          [=] __device__(int i) {
            auto tmp_index = dst_ids_ptr[i];
            return dst_ptr + dst_ev_start_indices_ptr[tmp_index];
          });

      multi_to_one_reduce_v2(multi_to_one_desc_first_stage, reduction_indices, kernel_params,
                             partial_reduce_result, wgrad, src_buffer.attr.max_ev_size, stream);
    });
  });
}

void LocalReduce::local_reduce(const ReductionIndices& reduction_indices,
                               const EmbeddingOutput& src_buffer, Wgrad& wgrad,
                               const core23::Tensor& local_lookup_ids, int num_lookup,
                               int num_global_lookup, int batch_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();
  int batch_size_per_gpu = batch_size / core_->get_global_gpu_count();

  HCTR_LIB_THROW(hipMemsetAsync(wgrad.data.data(), 0, wgrad.data.num_bytes(), stream));

  if (src_buffer.attr.layout == EmbeddingLayout::FeatureMajor) {
    dp_local_reduce_from_feature_major_top_grad(
        core_->get_kernel_param(), reduction_indices, src_buffer, local_lookup_ids, num_lookup,
        wgrad, partial_reduce_result_, batch_size_per_gpu, src_buffer.attr.max_ev_size, stream);
  } else {
    dp_local_reduce_from_batch_major_top_grad(core_->get_kernel_param(), reduction_indices,
                                              src_buffer, local_lookup_ids, num_global_lookup,
                                              wgrad, partial_reduce_result_, batch_size_per_gpu,
                                              src_buffer.attr.max_ev_size, stream);
  }
}

template <typename src_emb_t, typename dst_emb_t, typename offset_t>
struct DenseModelBackwardOneToOneAtomicDesc {
  using SrcT = src_emb_t;
  using DstT = dst_emb_t;
  HOST_DEVICE_INLINE int num_vec() { return num_vec_; }

  HOST_DEVICE_INLINE int get_vec_length(int i) { return ev_size; }
  // we need a transform to src id use num_model_revers_idx
  HOST_DEVICE_INLINE const SrcT* get_src_ptr(int i) { return src_ptr + i * ev_size; }
  HOST_DEVICE_INLINE DstT* get_dst_ptr(int i) { return dst_ptr + ev_size * reverse_id_ptr[i]; }

  size_t num_vec_;
  int ev_size;
  const offset_t* __restrict__ reverse_id_ptr;
  const src_emb_t* __restrict__ src_ptr;
  dst_emb_t* __restrict__ dst_ptr;
};

// for dense mp
void LocalReduce::local_reduce(const DenseReductionIndices& reduction_indices,
                               const DenseModelCommBuffer& src_buffer, Wgrad& wgrad) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  // int batch_size_per_gpu = batch_size / model_comm_buffer.attr.num_gpus;
  auto stream = core_->get_local_gpu()->get_stream();

  DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(src_buffer.attr.type.type(), src_emb_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(wgrad.attr.type.type(), dst_emb_t, [&] {
      DISPATCH_INTEGRAL_FUNCTION_CORE23(
          reduction_indices.model_reverse_idx->data_type().type(), offset_t, [&] {
            size_t num_keys = reduction_indices.reverse_key_num;
            int ev_size = reduction_indices.ev_size;
            offset_t* reverse_idx_ptr = reduction_indices.model_reverse_idx->data<offset_t>();
            src_emb_t* src_ptr = (src_emb_t*)src_buffer.data.data();
            dst_emb_t* dst_ptr = (dst_emb_t*)wgrad.data.data();
            using CopyDesc = DenseModelBackwardOneToOneAtomicDesc<src_emb_t, dst_emb_t, offset_t>;
            CopyDesc one_to_one_atomic_desc = {num_keys, ev_size, reverse_idx_ptr, src_ptr,
                                               dst_ptr};
            HCTR_LIB_THROW(hipMemsetAsync(
                wgrad.data.data(), 0,
                reduction_indices.num_valid_dst_tensor * ev_size * wgrad.data.data_type().size(),
                stream));
            one_to_one_atomic(one_to_one_atomic_desc, core_->get_kernel_param(), ev_size, num_keys,
                              stream);
          });
    });
  });
}

}  // namespace embedding
